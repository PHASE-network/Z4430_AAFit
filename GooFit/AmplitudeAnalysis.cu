#include "hip/hip_runtime.h"
#include "Variable.hh"
#include "ThreeBodiesPsiPiKPdf.hh"
#include "FitManager.hh"
#include "UnbinnedDataSet.hh"
#include "BinnedDataSet.hh"
#include "InterHistPdf.hh"
#include "FlatHistoPdf.hh"
#include "AddPdf.hh"
#include "ProdPdf.hh"
#include "MatrixPdf.hh"

#include "TNtupleD.h"
#include "TTree.h"
#include "TCanvas.h"
#include "TGraph.h"
#include "TH1F.h"
#include "TH2F.h"
#include "TString.h"
#include "TH1.h"
#include "TAxis.h"
#include "TROOT.h"
#include "TFile.h"
#include "TMath.h"
#include "TRandom.h"
#include "TMultiGraph.h"
#include "TPaveText.h"
#include "TAttLine.h"
#include "TGraph2D.h"

#include "BiDimHistoPdf.hh"

#include "../utilities.h"
// #include "../Angles_contour.h"
// #include "../Dalitz_contour.h"
// #include "../effMasses.h"

#include <vector>
#include <string>
#include <map>
#include <math.h>
#include <sstream>
#include <utility> // std::make_pair
#include <fstream>
#include "TLegend.h"

#include <sys/time.h> // for timeval
#include <sys/times.h> // for tms
#include <iostream>
#include "TSystem.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

//#define CUDADEBUGGING 50


timeval startTime, stopTime, totalTime;
clock_t startC, stopC;
tms startProc, stopProc;

const fptype M892 = 0.89581 ; const fptype G892 = 0.0474; // From PDG charged only K*(892)
const fptype M892e = 0.8961 ; const fptype G892e = 0.0507; // From EvtGen
const fptype M1410 = 1.414; const fptype G1410 = 0.232; // K*1410
const fptype M800 = 0.682; const fptype G800 = 0.547; //const fptype M800 = 0.931; const fptype G800 = 0.578; // K*800 Belle values: M = 0.946, G = 736
const fptype M1430_0 = 1.425; const fptype G1430_0 = 0.270; // K*1430_0
const fptype M1430_2 = 1.4324; const fptype G1430_2 = 0.109; // K*1430_2
const fptype M1780_3 = 1.776; const fptype G1780_3 = 0.159; // K*1780_3

const fptype TMATH_PI = TMath::Pi();

std::string migrad("MIGRAD"); std::string m("M");
std::string hesse("HESSE");   std::string h("H");
std::string minos("MINOS");   std::string n("N");

fptype phaseSpaceFunction(fptype x,fptype mP,fptype m1,fptype m2,fptype m3)
{
  fptype function = isnan(sqrt(pow(x,4) + pow(m1,4) + pow(m2,4) - 2*pow(x,2)*pow(m1,2) - 2*pow(x,2)*pow(m2,2) - 2*pow(m1,2)*pow(m2,2)) * sqrt(pow(mP,4) + pow(x,4) + pow(m3,4) - 2*pow(mP,2)*pow(x,2) - 2*pow(mP,2)*pow(m3,2) - 2*pow(x,2)*pow(m3,2) ) / (x)) ? 0 : (sqrt(pow(x,4) + pow(m1,4) + pow(m2,4) - 2*pow(x,2)*pow(m1,2) - 2*pow(x,2)*pow(m2,2) - 2*pow(m1,2)*pow(m2,2)) * sqrt(pow(mP,4) + pow(x,4) + pow(m3,4) - 2*pow(mP,2)*pow(x,2) - 2*pow(mP,2)*pow(m3,2) - 2*pow(x,2)*pow(m3,2) ) / (x));

  return function;
}

void debug(int line) {
  #ifdef CUDADEBUGGING
  std::cout <<"Debugging on line " <<line <<std::endl;
  #endif
}

int parDotSpin (fptype dotSpin) {
  int result = static_cast<int>(floor((dotSpin - floor(dotSpin))*10.+.1));
  return result;
}

std::string doubleToStr (fptype dbl) {
  std::ostringstream strs;
  strs << dbl;
  return strs.str();
}

Int_t compBins = 50, pdfBins = 50, dataBins = 100;
//Int_t compBins = 100, pdfBins = 100, dataBins = 100;

void addHelAmplStat(TPaveText *fitStat, TString hel, Variable* a, Variable* b) {
  TString a_value = TString::Format("a_{%s} = %.2f",hel.Data(),a->value) ;
  TString a_error = TString::Format("#pm %.2f",a->error) ;
  if (a->fixed) a_error = "fixed";

  TString b_value = TString::Format("b_{%s} = %.2f",hel.Data(),b->value) ;
  TString b_error = TString::Format("#pm %.2f",b->error) ;
  if (b->fixed) b_error = "fixed";

  fitStat->AddText(TString::Format("%s %s, %s %s",a_value.Data(),a_error.Data(),b_value.Data(),b_error.Data()));
}

void printinstruction() {

  std::cerr << "======= Instructions \n"
  	        << "\t-h,--help \t\t Show this help message\n"
            << "\t-evtGen \t\t Select EvtGen dataset and p.d.f. parameters\n"
            << "\t-effH <4-dig-code> \t Perform the product of the pdf by the efficiency histogram ()\n"
            << "\t\t\t\t\t\t - code1 ()\n"
	          << "\t-n <events> \t\t Specify the number of events to use\n"
  	        << "\t-r <path> \t\t Read Generated Events from txt in <path>\n"
            << "\t-algos <algo1algo2...>\t Select the mimimisation algos in the order they should \n \t\t\t\tbe performed (MIGRAD at least once) ["<<m<<" for MIGRAD, "<<h<<" for HESSE, "<<n<<" for MINOS]\n \t\t\t\t (e.g -algo "<<h<<m<<h<<n<<" for HESSE MIGRAD HESSE MINOS - default: MIGRAD only) \n"
  	        << "\t-b1 <b1> \t\t Select binning for massKPi (for normalisation & integration, default: 40)\n"
            << "\t-b2 <b2> \t\t Select binning for CosMuMu (for normalisation & integration, default: 40)\n"
            << "\t-b3 <b3> \t\t Select binning for massPsiPi (for normalisation & integration, default: 40)\n"
            << "\t-b4 <b4> \t\t Select binning for Phi (for normalisation & integration, default: 40)\n"
            << "\t-p1 <p> \t\t Select p.d.f. plotting binning finenness (default: " <<pdfBins <<") for MassKPi \n"
            << "\t-p2 <p> \t\t Select p.d.f. plotting binning finenness (default: " <<pdfBins <<") for CosMuMu \n"
            << "\t-p3 <p> \t\t Select p.d.f. plotting binning finenness (default: " <<pdfBins <<") for MassPsiPi \n"
            << "\t-p4 <p> \t\t Select p.d.f. plotting binning finenness (default: " <<pdfBins <<") for Phi \n"
            << "\t-d1 <p> \t\t Select dataset binning (default: " <<dataBins <<") for MassKPi \n"
            << "\t-d2 <p> \t\t Select dataset binning (default: " <<dataBins <<") for CosMuMu \n"
            << "\t-d3 <p> \t\t Select dataset binning (default: " <<dataBins <<") for MassPsiPi \n"
            << "\t-d4 <p> \t\t Select dataset binning (default: " <<dataBins <<") for Phi \n"
            << "\t-Bb <Bb> \t\t Select bound limits for b parameter (default: 9999)\n"
            << "\t-k800 \t\t\t Add K*_0(800) to p.d.f.\n"
            << "\t-k892 \t\t\t Add K*_1(892) to p.d.f.\n"
            << "\t-k1410 \t\t\t Add K*_1(1410) to p.d.f.\n"
            << "\t-k1430_0 \t\t Add K*_0(1430) to p.d.f.\n"
            << "\t-k1430_2 \t\t Add K*_2(1430) to p.d.f.\n"
            << "\t-k1780 \t\t\t Add K*_3(1780) to p.d.f.\n"
            << "\t-BkgMap \t\t Add Phase Space Background Map \n"
            << "\t-Bkg \t\t\t Add Phase Space Background to p.d.f.\n"
            << std::endl;
}


int main(int argc, char** argv) {
  debug(__LINE__);

  char bufferstring[1024];

  unsigned int events = 1000000;
  unsigned int nKstars = 0;

  unsigned int bin1 = compBins, bin2 = compBins, bin3 = compBins, bin4 = compBins;
  unsigned int datapoints1 = dataBins, datapoints2 = dataBins, datapoints3 = dataBins, datapoints4 = dataBins;
  unsigned int plottingfine1 = pdfBins, plottingfine2 = pdfBins, plottingfine3 = pdfBins, plottingfine4 = pdfBins;

  fptype aMax = +9999.;
  fptype bMax = +9999.;

  bool k892Star = false;
  bool k800Star = false;
  bool k1410Star = false;
  bool k1430Star0 = false;
  bool k1430Star2 = false;
  bool k1780Star = false;

  bool bkgPhaseSpace = false;
  bool bkgPhaseSpaceMap = false;
  bool effPdfProd = false;

  bool evtGen = false;

  bool txtfile = false;
  //bool hesse = false;

  std::vector<std::string> algos;
  algos.push_back(migrad);

  TString datasetName = "Kstars";
  std::string underscores = "__";
  TString plotsDir = "./plots";
  std::vector< std::string> kStarNames;

  TH2F* relEffTH2Mass = 0;
  TH2F* relEffTH2Ang = 0;
  TH2F* bkgTH2Mass = 0, *bkgTH2Ang = 0;

  TH1D* bkgMKPi=0, *bkgMPsiPi=0, *bkgPhi=0, *bkgCMuMu = 0;

    if (argc<=1)
      {
        printinstruction();
        return 0;
      }

    for (int i = 1; i < argc; ++i)
      {
        std::string arg = argv[i];
        if ((arg == "-h") || (arg == "--help"))
    	{
    	  printinstruction();
    	  return 0;
    	}
        else if (arg == "-n")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> events))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-b1")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> bin1))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-b2")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> bin2))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-b3")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> bin3))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-b4")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> bin4))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-p1")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> plottingfine1))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-p2")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> plottingfine2))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-p3")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> plottingfine3))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-p4")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> plottingfine4))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-Bb")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> bMax))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	  else
  	    bMax = TMATH_PI;
  	}
        else if (arg == "-k892")
  	{
  	  k892Star = true;
  	  ++nKstars;
  	}
        else if (arg == "-k800")
  	{
  	  k800Star = true;
  	  ++nKstars;
  	}
        else if (arg == "-k1410")
  	{
  	  k1410Star = true;
  	  ++nKstars;
  	}
        else if (arg == "-k1430_0")
  	{
  	  k1430Star0 = true;
  	  ++nKstars;
  	}
        else if (arg == "-k1430_2")
  	{
  	  k1430Star2 = true;
  	  ++nKstars;
  	}
        else if (arg == "-k1780")
  	{
  	  k1780Star = true;
  	  ++nKstars;
  	}
    else if (arg == "-Bkg")
    {
      bkgPhaseSpace = true;
      bkgPhaseSpaceMap = false;
    }
    else if (arg == "-BkgMap")
    {
      bkgPhaseSpaceMap = true;
      bkgPhaseSpace = false;
    }
    else if (arg == "-effH")
    {
      effPdfProd = true;
    }
    else if (arg == "-txt")
    {
      txtfile = true;
    }
        else if (arg == "-d1")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> datapoints1))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-d2")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> datapoints2))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-d3")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> datapoints3))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
        else if (arg == "-d4")
  	{
  	  if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;
  	      std::istringstream ss(argv[i]);
  	      if (!(ss >> datapoints4))
  		{
  		  std::cerr << "Invalid number " << argv[i] << '\n';
  		  exit(1);
  		}
  	    }
  	}
    //     else if (arg == "-H")
  	// {
  	//   hesse = true;
  	// }
        else if (arg == "-algos")
    {
      algos.clear();
      if (i + 1 < argc) // Make sure we aren't at the end of argv!
  	    {
  	      i++;

  	      std::string algosInput= argv[i];
          std::size_t found = algosInput.find(m);

          if (found == std::string::npos)
          {
            std::cout << "Minimisation algorithms invalid input : MIGRAD to be called at least once \n";
            exit(1);
          }
          std::cout << "- Minimisation algorithms sequence : "<<std::endl;

          for (std::string::size_type l = 0; l < algosInput.length(); ++l)
          {
            std::string::value_type algo = algosInput[l];

            if(algo==m)
            {
              algos.push_back(migrad);
              std::cout<<"  - MIGRAD "<<std::endl;
            }
            else if(algo==h)
            {
              algos.push_back(hesse);
              std::cout<<"  - HESSE "<<std::endl;
            }
            else if(algo==n)
            {
              algos.push_back(minos);
              std::cout<<"  - MINOS "<<std::endl;
            }
            else std:: cout<<"  - \""<<algo<<"\" invalid input, ignored "<<std::endl;
          }

  	    }
      }


    }

      if (bin1 > plottingfine1)
        cout <<"WARNING! Bins for normalisation & integration (" <<bin1 <<") are more than bins for p.d.f. plotting (" <<plottingfine1 <<")\n" <<endl;
      if (bin2 > plottingfine2)
        cout <<"WARNING! Bins for normalisation & integration (" <<bin2 <<") are more than bins for p.d.f. plotting (" <<plottingfine2 <<")\n" <<endl;
      if (bin3 > plottingfine3)
        cout <<"WARNING! Bins for normalisation & integration (" <<bin3 <<") are more than bins for p.d.f. plotting (" <<plottingfine3 <<")\n" <<endl;
      if (bin4 > plottingfine4)
        cout <<"WARNING! Bins for normalisation & integration (" <<bin4 <<") are more than bins for p.d.f. plotting (" <<plottingfine4 <<")\n" <<endl;

  TString plotsName = "";
  TString extension = "eps"; extension = "png";

  if (!nKstars) {
    cout <<"No K* selected (K892,K800,K1410,K1430) please see instructions below" <<endl;
    printinstruction();
    return 1;
  } else {
    cout <<"- Performing Amplitude Analysis fit with\n  " <<nKstars <<" K*(s) on\n  " <<events <<" events, using\n  " <<bin1 <<" bins for normalisation & integration and\n  " <<plottingfine1 <<" bins for p.d.f. plotting" <<endl;
    if (nKstars < 2) {
      datasetName = "Kstar";
      underscores = "_"; }

    if (k892Star) {
      cout <<"  - K*(892)" <<endl;
      datasetName.Append(underscores+"892_1"); plotsName.Append("__892_1");
      kStarNames.push_back("K*_{1}(892)");
    }
    if (k800Star) {
      cout <<"  - K*(800)" <<endl;
      datasetName.Append(underscores+"800_0"); plotsName.Append("__800_0");
      kStarNames.push_back("K*_{0}(800)");
    }
    if (k1410Star) {
      cout <<"  - K*(1410)" <<endl;
      datasetName.Append(underscores+"1410_1"); plotsName.Append("__1410_1");
      kStarNames.push_back("K*_{1}(1410)");
    }
    if (k1430Star0) {
      cout <<"  - K*(1430_0)" <<endl;
      datasetName.Append(underscores+"1430_0"); plotsName.Append("__1430_0");
      kStarNames.push_back("K*_{0}(1430)");
    }
    if (k1430Star2) {
      cout <<"  - K*(1430_2)" <<endl;
      datasetName.Append(underscores+"1430_2"); plotsName.Append("__1430_2");
      kStarNames.push_back("K*_{2}(1430)");}
    if (k1780Star) {
      cout <<"  - K*(1780_3)" <<endl;
      datasetName.Append(underscores+"1780_3"); plotsName.Append("__1780_3");
      kStarNames.push_back("K*_{3}(1780)");}
    if (bkgPhaseSpace) {
      cout <<"  - Three Bodies Phase-space background" <<endl;
      datasetName.Append("__plus__BdToPsiPiK_PHSP"); plotsName.Append("_PHSP");
    }
    if (bkgPhaseSpaceMap) {
      cout <<"  - Combinatorial background" <<endl;
      datasetName.Append("__plus__BdToPsiPiK_combinatorial"); plotsName.Append("__combinatorialBkg");
    }
    if (effPdfProd) {
      cout <<"  - With efficiency multiplication" <<endl;
      plotsName.Append("__effCorr");
    }
  }

  fptype aMin = -aMax;
  fptype bMin = -bMax;

  debug(__LINE__);

  //CANVAS
  TCanvas* canvas = new TCanvas("","",2000,1200);

  Variable* dRadB0  = new Variable("dRadB0",5.0);
  Variable* dRadKs  = new Variable("dRadKs",1.5);
  Variable* psi_nS  = new Variable("psi_nS",1.0);

  //std::vector<Variable* > amplitudeGooVars;
  //std::vector<Variable*> KParams;

  //GooFit
  Variable mBd("mBd", 5.27961) ;
  Variable mKaon("mKaon", 0.493677) ;
  Variable mPion("mPion", 0.13957018) ;

  fptype massMuMu = 0. ;
  if (psi_nS->value==1.0) massMuMu = 3.096916 ;
  else if (psi_nS->value==2.0) massMuMu = 3.686109 ;
  else {
    cout <<"psi_nS is neither 1 nor 2, please check it." <<endl;
    return 1; }
  Variable mMuMu("mMuMu", massMuMu);
  const fptype smearing = 0. ;
  Variable smear("smear",smearing) ;
  debug(__LINE__);

  //TH1F* dataHisto = new TH1F("data","data",BINS,massKPi.lowerlimit,massKPi.upperlimit);
  //TH1F pdfBkgHist ("bkg","bkg",BINS,massKPi.lowerlimit,massKPi.upperlimit);
  // B^{0} -> psi(nS) #pi^{+} K^{-}

  //cout <<"\nBdToMuMuPiK_PHSP.getVal() =\n" <<BdToMuMuPiK_PHSP->getVal() <<endl; return;

  //fptype roll=0.0;
  //fptype func=0.0;

  long int ms; struct timeval tp;

  gettimeofday(&tp,NULL);
  ms = tp.tv_sec * 1000 + tp.tv_usec / 1000;
  TRandom ranGen(ms);
  debug(__LINE__);
  /*
    for (int j = 0; j < events; ++j) {

    massKPi.value = ranGen.Uniform(massKPi.upperlimit - massKPi.lowerlimit) + massKPi.lowerlimit;
    func = phaseSpaceFunction(massKPi.value,MBd,MPion,MKaon,massMuMu);
    roll = ranGen.Uniform(100);
    if (roll > func) {
    --j;
    continue; }

    if ((massKPi.value < massKPi.lowerlimit) || (massKPi.value > massKPi.upperlimit)) {
    --j;
    continue;}

    dataHisto->Fill(massKPi.value);

    }


    for (int i = 1; i < BINS; i++) {
    dataSet.setBinContent(i-1,dataHisto->GetBinContent(i));
    }

    debug(__LINE__);
    background->setData(&dataSet);
    background->setFitControl(new BinnedNllFit());

    FitManager fitterNull(background);
    fitterNull.fit();
    fitterNull.getMinuitValues();

    vector<double> ValsFondo;
    debug(__LINE__);
    background->evaluateAtPoints(&massKPi,ValsFondo);
    fptype totalFondo=0.0;
    for (int k=0;k<BINS;k++) {

    pdfBkgHist.SetBinContent(k+1,ValsFondo[k]);
    totalFondo += ValsFondo[k];
    std::cout << ValsFondo[k]<<std::endl;

    }
    debug(__LINE__);
    for (int k=0;k<BINS;k++) {
    fptype valTot = pdfBkgHist.GetBinContent(k+1);
    valTot /= totalFondo;
    valTot *= events;
    pdfBkgHist.SetBinContent(k+1, valTot);
    cout <<" " <<pdfBkgHist.GetBinContent(k+1)<<endl;
    }
    debug(__LINE__);
    pdfBkgHist.SetFillStyle(3002);
    pdfBkgHist.SetFillColor(kGreen);

    double likeliHoodNull = 0.0;
    double likeliHoodSignal = 0.0;

    debug(__LINE__);
    TCanvas canvas("canvas","canvas",1000,1000);

    dataHisto->Draw();
    pdfBkgHist.Draw("same");

    canvas.SaveAs("./plots/test.png");
  */

  //Defining minimums and maximums
  fptype massKPi_min = 0.6, massKPi_max = 2.2;
  fptype massPsiPi_min = 3.2, massPsiPi_max = 4.9;


  TString massKPi_name = "massKPi", cosMuMu_name = "cosMuMu", massPsiPi_name = "massPsiPi", phi_name = "phi";
  TString massKPi_eff_name = "massKPiEff", massPsiPi_eff_name = "massPsiPiEff";
  Variable* massKPi = new Variable(massKPi_name.Data(),1.,massKPi_min,massKPi_max); massKPi->numbins = bin1;
  Variable* massKPiEff = new Variable(massKPi_eff_name.Data(),1.,0.6,2.2); massKPiEff->numbins = bin1;
  //Variable* massKPi = new Variable(massKPi_name.Data(),1.,0.6,1.67); massKPi->numbins = bin1;
  Variable* massPsiPi = new Variable(massPsiPi_name.Data(),TMath::Sqrt(23),massPsiPi_min,massPsiPi_max); massPsiPi->numbins = bin3;
  Variable* massPsiPiEff = new Variable(massPsiPi_eff_name.Data(),TMath::Sqrt(23),3.2,4.9); massPsiPiEff->numbins = bin3;
  // cosine of the psi(nS) helicity angle
  Variable* cosMuMu = new Variable(cosMuMu_name.Data(),0.,-1,1); cosMuMu->numbins = bin2;
  // cosine of the K* helicity angle
  //Variable* massPsiPi = new Variable(massPsiPi_name.Data(),0.,-1,1); massPsiPi->numbins = bin3;
  // angle between decay planes
  Variable* phi = new Variable(phi_name.Data(),0.25,-TMATH_PI,TMATH_PI); phi->numbins = bin4;

  //fptype ratio = ((fptype)(plottingfine))/((fptype)massKPi->numbins);
  fptype ratioMKPi = ((fptype)(plottingfine1))/((fptype)datapoints1);
  fptype ratioCosMuMu = ((fptype)(plottingfine2))/((fptype)datapoints2);
  fptype ratioMassPsiPi = ((fptype)(plottingfine3))/((fptype)datapoints3);
  fptype ratioPhi = ((fptype)(plottingfine4))/((fptype)datapoints4);

  std::vector<Variable*> obserVariables;
  obserVariables.push_back(massKPi);
  obserVariables.push_back(massPsiPi);
  obserVariables.push_back(cosMuMu);
  obserVariables.push_back(phi);

  std::vector<Variable*> massVariable;
  massVariable.push_back(massKPi);

  std::vector<Variable*> massesVariables;
  massesVariables.push_back(massKPi);
  massesVariables.push_back(massPsiPi);

  // std::vector<Variable*> obserMasses;
  // obserMasses.push_back(massKPi);
  // obserMasses.push_back(massPsiPi);


  std::vector<Variable*> Masses, Gammas, Spins, as, bs;

  if (k892Star) {
    cout <<"\nAdding K*(892) ..." <<endl;

    if (!evtGen) {
      Masses.push_back(new Variable("K_892_Mass_0",M892));
      Gammas.push_back(new Variable("K_892_Gamma_0",G892));
      Spins.push_back(new Variable("K_892_Spin_0",1.0));
      as.push_back(new Variable("a_K_892_0",1.0));//,aMin,aMax) );
      bs.push_back(new Variable("b_K_892_0",0.0));//,bMin,bMax) );
      as.push_back(new Variable("a_K_892_p1",0.844,aMin,aMax) );
      bs.push_back(new Variable("b_K_892_p1",3.14,bMin,bMax) );
      as.push_back(new Variable("a_K_892_m1",0.196,aMin,aMax));
      bs.push_back(new Variable("b_K_892_m1",-1.7,bMin,bMax));
    } else {
      Masses.push_back(new Variable("K_892_Mass_0",M892e));
      Gammas.push_back(new Variable("K_892_Gamma_0",G892e));
      Spins.push_back(new Variable("K_892_Spin_0",1.0));
      // EvtGen
      as.push_back(new Variable("a_K_892_0",0.775));
      //bs.push_back(new Variable("b_K_892_0",0.0));
      //as.push_back(new Variable("a_K_892_0",0.775,0.50,0.8));
      bs.push_back(new Variable("b_K_892_0",0.0));
      as.push_back(new Variable("a_K_892_p1",0.159,0.14,0.17) );
      bs.push_back(new Variable("b_K_892_p1",1.563,1.4,1.57) );
      as.push_back(new Variable("a_K_892_m1",0.612,0.50,0.63));
      bs.push_back(new Variable("b_K_892_m1",2.712,1.0,2.73));
    }

  }

  if (k800Star) {
    cout <<"Adding K*(800) ..." <<endl;

    Masses.push_back(new Variable("K_800_Mass_0",M800));
    Gammas.push_back(new Variable("K_800_Gamma_0",G800));
    Spins.push_back(new Variable("K_800_Spin_0",0.0));
    as.push_back(new Variable("a_K_800_0",1.12,aMin,aMax) );
    bs.push_back(new Variable("b_K_800_0",2.3,bMin,bMax) );
  }

  if (k1410Star) {
    cout <<"Adding K*(1410) ..." <<endl;

    Masses.push_back(new Variable("K_1410_Mass_0",M1410));
    Gammas.push_back(new Variable("K_1410_Gamma_0",G1410));
    Spins.push_back(new Variable("K_1410_Spin_0",1.0));
    as.push_back(new Variable("a_K_1410_0",0.119,aMin,aMax) );
    bs.push_back(new Variable("b_K_1410_0",0.81,bMin,bMax) );

    //as.push_back(new Variable("a_K_1410_0",0.844));
    //bs.push_back(new Variable("b_K_1410_0",3.14,bMin,bMax));

    as.push_back(new Variable("a_K_1410_p1",0.123,aMin,aMax) );
    bs.push_back(new Variable("b_K_1410_p1",-1.04,bMin,bMax) );

    as.push_back(new Variable("a_K_1410_m1",0.036,aMin,aMax));
    bs.push_back(new Variable("b_K_1410_m1",0.67,bMin,bMax));
  }

  if (k1430Star0) {
    cout <<"Adding K*(1430_0) ..." <<endl;

    Masses.push_back(new Variable("K_1430_0_Mass_0",M1430_0));
    Gammas.push_back(new Variable("K_1430_0_Gamma_0",G1430_0));
    Spins.push_back(new Variable("K_1430_0_Spin_0",0.0));
    as.push_back(new Variable("a_K_1430_0_0",0.89,aMin,aMax) );
    bs.push_back(new Variable("b_K_1430_0_0",-2.17,bMin,bMax) );
  }

  if (k1430Star2) {
    cout <<"Adding K*(1430_2) ..." <<endl;

    Masses.push_back(new Variable("K_1430_2_Mass_0",M1430_2));
    Gammas.push_back(new Variable("K_1430_2_Gamma_0",G1430_2));
    Spins.push_back(new Variable("K_1430_2_Spin_0",2.0));
    as.push_back(new Variable("a_K_1430_2_0",4.66,aMin,aMax) );
    bs.push_back(new Variable("b_K_1430_2_0",-0.32,bMin,bMax) );

    //as.push_back(new Variable("a_K_1430_2_0",0.844));
    //bs.push_back(new Variable("b_K_1430_2_0",3.14,bMin,bMax));

    as.push_back(new Variable("a_K_1430_2_p1",4.65,aMin,aMax) );
    bs.push_back(new Variable("b_K_1430_2_p1",-3.05,bMin,bMax) );

    as.push_back(new Variable("a_K_1430_2_m1",1.26,aMin,aMax));
    bs.push_back(new Variable("b_K_1430_2_m1",-1.92,bMin,bMax));
  }

  if (k1780Star) {
    cout <<"Adding K*(1780)_3 ..." <<endl;

    Masses.push_back(new Variable("K_1780_3_Mass_0",M1780_3));
    Gammas.push_back(new Variable("K_1780_3_Gamma_0",G1780_3));
    Spins.push_back(new Variable("K_1780_3_Spin_0",3.0));
    as.push_back(new Variable("a_K_1780_3_0",16.8,aMin,aMax) );
    bs.push_back(new Variable("b_K_1780_3_0",-1.43,bMin,bMax) );

    //as.push_back(new Variable("a_K_1780_3_0",0.844));
    //bs.push_back(new Variable("b_K_1780_3_0",3.14,bMin,bMax));

    as.push_back(new Variable("a_K_1780_3_p1",19.1,aMin,aMax) );
    bs.push_back(new Variable("b_K_1780_3_p1",2.03,bMin,bMax) );

    as.push_back(new Variable("a_K_1780_3_m1",10.2,aMin,aMax));
    bs.push_back(new Variable("b_K_1780_3_m1",1.55,bMin,bMax));
  }

  Int_t nHelAmps = as.size();

  //DATASET

  UnbinnedDataSet dataset(obserVariables);
  BinnedDataSet masseKPiDataset(massKPi);
  BinnedDataSet massesDataset(massesVariables);
  UnbinnedDataSet masseKPiDatasetUn(massKPi);

  std::cout<<"Dataset : "<<std::endl;
  // std::cout<<" - dataset with "<<dataset.getNumBins()<<" bins "<<std::endl;
  // std::cout<<" - massesDataset with "<<massesDataset.getNumBins()<<" bins "<<std::endl;
  //std::cout<<" - efficiencyDatasetMasses with "<<efficiencyDatasetMasses->getNumBins()<<" bins "<<std::endl;

  //TString massKPi_title = "m(K^{-}#pi^{+})",  cosMuMu_title = "cos(#theta_{J/#psi})",  massPsiPi_title = "cos(#theta_{K*})",  phi_title = "#phi";
  TString massKPi_title = "m(K^{-}#pi^{+})",  cosMuMu_title = "cos(#theta_{J/#psi})",  massPsiPi_title = "m(J/#psi#pi^{+})",  phi_title = "#phi";
  TH1F massKPiHisto(massKPi_name+"_Histo", TString::Format("%s;%s [GeV]",massKPi_name.Data(),massKPi_title.Data()), datapoints1, massKPi->lowerlimit, massKPi->upperlimit); massKPiHisto.SetLineColor(kBlack); massKPiHisto.SetMarkerColor(kBlack);
  TH1F cosMuMuHisto(cosMuMu_name+"_Histo", TString::Format("%s;%s",cosMuMu_name.Data(),cosMuMu_title.Data()), datapoints2, cosMuMu->lowerlimit, cosMuMu->upperlimit); cosMuMuHisto.SetLineColor(kBlack); cosMuMuHisto.SetMarkerColor(kBlack);
  TH1F massPsiPiHisto(massPsiPi_name+"_Histo", massPsiPi_name+";"+massPsiPi_title+" [GeV]", datapoints3, massPsiPi->lowerlimit, massPsiPi->upperlimit); massPsiPiHisto.SetLineColor(kBlack); massPsiPiHisto.SetMarkerColor(kBlack);
  TH1F phiHisto(phi_name+"_Histo", phi_name+";"+phi_title, datapoints4, phi->lowerlimit, phi->upperlimit); phiHisto.SetLineColor(kBlack); phiHisto.SetMarkerColor(kBlack);

  TString path;
  path = "/lustrehome/cristella/work/Z_analysis/exclusive/clean_14ott/original/CMSSW_5_3_22/src/UserCode/MuMuPiKPAT/test/sanjay/selector/TMVA/";

  //datasetName = "dataGen_B0"; //datasetName = "dataGen_B0bar";
  //datasetName.Append("_B0massConstraint");
  if (datasetName.Contains("dataGen_B0")) plotsDir.Append("/B0");
  else if (datasetName.Contains("dataGen_B0bar")) plotsDir.Append("/B0bar");

  if(evtGen) datasetName.Append("__EvtGen");
  //datasetName.Append("_mPhi");
  if (txtfile) datasetName.Append(".txt");
  TString fullDatasetName = "./datasets/"+datasetName;
  fullDatasetName = "../datasets/"+datasetName;

  if (txtfile) {
    ifstream dataTxt(fullDatasetName.Data());
    Int_t totEvents = 0;
    if ( !(dataTxt.good()) ) {
      std::cout <<"No valid input at : " <<fullDatasetName <<" provided.\nReturning." <<std::endl;
      return 1;
    } else {
      totEvents = std::count(std::istreambuf_iterator<char>(dataTxt), std::istreambuf_iterator<char>(), '\n');
      if (events > totEvents) {
	cout <<"\nWARNING! The number of events requested is " <<events <<" but " <<fullDatasetName <<" contains only " <<totEvents <<" events." <<endl;
	events = totEvents;
      }

      fptype var1, var2, var3, var4;

      Int_t evt=0;
      cout <<"\n- Reading " <<events <<" out of " <<totEvents <<" events from " <<datasetName <<" and filling variables histograms" <<endl;
      dataTxt.clear(); dataTxt.seekg (0, ios::beg);
      while( (evt < events)  &&  (dataTxt >> var1 >> var2 >> var3 >> var4) ) {
	evt++;
	massKPi->value = var1;
	massPsiPi->value = var2;
	cosMuMu->value = var3;
	phi->value = var4;

	//std::cout << massKPi->value << " - " <<cosMuMu->value << " - " << massPsiPi->value << " - " << phi->value << " - " << std::endl;
	if (Dalitz_contour_host(massKPi->value, massPsiPi->value, kFALSE, (Int_t)psi_nS->value) ) {
	  dataset.addEvent();
	  masseKPiDatasetUn.addEvent();
	  massesDataset.addEvent();
	  //if(massesDataset.getNumEvents()==0) massesDataset.addEvent();
	  massKPiHisto.Fill(massKPi->value);
	  cosMuMuHisto.Fill(cosMuMu->value);
	  massPsiPiHisto.Fill(massPsiPi->value);
	  phiHisto.Fill(phi->value);
	}

	dataTxt.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
      }
    }
    dataTxt.close();
  } // if (txtfile)
  else {
    //TString dataFileName = "./datafiles/Data_JPsi_2p0Sig_6p0to9p0SB.root";
    //TString dataFileName = "./datafiles/TMVApp_withBDTCutAt0p00_JPsi_2p0Sig_6p0to9p0SB.root";
    // path = "./datafiles/";
    TString dataFileName = path+"TMVApp_data_withBDTCutAt0p00_JPsi_2p0Sig_6p0to9p0SB.root";
    TFile *inputFile = TFile::Open(dataFileName);

    if (!inputFile) {
      cout <<"Warning: unable to open data file \"" <<dataFileName <<"\"" <<endl;
    } else {

      TString dataTreeName = "AAVars";
      TNtupleD* dataNTuple = (TNtupleD*)inputFile->Get(dataTreeName);

      if(!(dataNTuple)){
	std::cout<<"Data NTuple named \'"<<dataTreeName<<"\' not found in TFile \'" <<inputFile->GetName() <<"\'.\nReturning."<<std::endl;
	return -1;
      }

      Double_t obs1,obs2,obs3,obs4;

      dataNTuple->SetBranchAddress("massKPi",&obs1);
      dataNTuple->SetBranchAddress("massMuMuPi",&obs2);
      dataNTuple->SetBranchAddress("cosMuMu",&obs3);
      dataNTuple->SetBranchAddress("phi",&obs4);

      Int_t nEntries = dataNTuple->GetEntries();
      if (events > nEntries) {
	cout <<"\nWARNING! The number of events requested is " <<events <<" but " <<dataFileName <<" contains only " <<nEntries <<" events." <<endl;
	events = nEntries;
      }

      cout <<"\n- Reading " <<events <<" out of " <<nEntries <<" events from " <<dataFileName <<" and filling variables histograms" <<endl;
      for (Int_t i=0; i<events; i++) {
	dataNTuple->GetEntry(i);

	//std::cout<<obs1<<" - "<<obs2<<" - "<<obs3<<" - "<<obs4<<std::endl;
	massKPi->value = obs1;
	massPsiPi->value = obs2;
	cosMuMu->value = obs3;
	phi->value = obs4;

	if (Dalitz_contour_host(massKPi->value, massPsiPi->value, kFALSE, (Int_t)psi_nS->value) )
	  {
	    dataset.addEvent();
	    masseKPiDatasetUn.addEvent();
	    massesDataset.addEvent();
	    //if(massesDataset.getNumEvents()==0) massesDataset.addEvent();
	    massKPiHisto.Fill(massKPi->value);
	    cosMuMuHisto.Fill(cosMuMu->value);
	    massPsiPiHisto.Fill(massPsiPi->value);
	    phiHisto.Fill(phi->value);
	  }
      }

    }
  } // if (!txtfile)


  for (size_t i = 0; i < massKPi->numbins; i++) {
    //masseKPiDataset.setBinContent(i,1000 - (500*i/massKPi->numbins));
    masseKPiDataset.setBinContent(i,massKPiHisto.GetBinContent(i+1));
  }

  if (dataset.getNumEvents() < 1) {
    cout <<"No events added from "  <<fullDatasetName <<"\nReturning." <<endl;
    return 0;
  } else
    std::cout <<"Added " <<dataset.getNumEvents() <<" events within Dalitz border to GooFit dataset" <<std::endl;

  events = dataset.getNumEvents();

  ////////////////////////////////////
  //Efficiencies

  GooPdf* efficiencyHistMasses;
  GooPdf* efficiencyHistAngles;
  GooPdf* effHist;
  BinnedDataSet* effDataset, *effDatasetMasses, *effDatasetAngles;

  ////////////////////////////////////
  //Backgrounds
  GooPdf* bkgHistMasses;
  GooPdf* bkgHistAngles;
  BinnedDataSet* bkgDatasetMasses;
  BinnedDataSet* bkgDatasetAngles;
  BinnedDataSet* bkgDataset;

  //BinnedDataSet effDatasetAngles(obserVariables,"efficiency Dataset Angles");
  std::cout<<"Initialising pdfs " <<std::endl;
  std::cout<<"- Matrix p.d.f. " <<std::endl;
  if (effPdfProd) {

    int iVar1 = 0, iVar2 = 1, iVar3 = 2, iVar4 = 3;
    int holdBinVar1, holdBinVar2,holdBinVar4,holdBinVar3;

    //int outCounter = 0;

    // path = "./effFiles/";
    //TString effName = "officialMC_noPtEtaCuts_JPsi_Bd2MuMuKPi_2p0Sig_4p0to6p0SB.root";
    TString effName = "TMVApp_MC_withBDTCutAt0p00_JPsi_2p0Sig_6p0to9p0SB.root";

    TFile *effFile = TFile::Open(path+effName);
    TString relEffNameMass = "RelEff_psi2SPi_vs_KPi_B0constr_1B0_BDTCutAt0p00";
    TString relEffNameAng = "RelEff_planesAngle_vs_cos_psi2S_helicityAngle_BDTCutAt0p00";

    // TString relEffNameMass = "RelEff_psi2SPi_vs_KPi_B0constr";
    // TString relEffNameAng = "RelEff_planesAngle_vs_cos_psi2S_helicityAngle";

    relEffTH2Mass = (TH2F*)effFile->Get(relEffNameMass) ;
    relEffTH2Ang = (TH2F*)effFile->Get(relEffNameAng) ;

    if(!(relEffTH2Mass)) {
      std::cout<<"Efficiency TH2 named \'"<<relEffNameMass<<"\' NOT FOUND in found in TFile \'" <<effFile->GetName() <<"\'.\nReturning."<<std::endl;
      return -1;
    }
    if(!(relEffTH2Ang)) {
      std::cout<<"Efficiency TH2 named \'"<<relEffNameAng<<"\' NOT FOUND in found in TFile \'" <<effFile->GetName() <<"\'.\nReturning."<<std::endl;
      return -1;
    }

    std::cout<<"Masses efficiency TH2 read with bin massKPi = " <<relEffTH2Mass->GetNbinsX() <<" and bin massPsiPi = " <<relEffTH2Mass->GetNbinsY() <<std::endl;
    std::cout<<"Angles efficiency TH2 read with bin x = " <<relEffTH2Ang->GetNbinsX() <<" and bin y = " <<relEffTH2Ang->GetNbinsY() <<std::endl;

    holdBinVar1 = obserVariables[iVar1]->numbins;
    holdBinVar2 = obserVariables[iVar2]->numbins;
    holdBinVar3 = obserVariables[iVar3]->numbins;
    holdBinVar4 = obserVariables[iVar4]->numbins;

    obserVariables[iVar1]->numbins = relEffTH2Mass->GetNbinsX();
    obserVariables[iVar2]->numbins = relEffTH2Mass->GetNbinsY();
    obserVariables[iVar3]->numbins = relEffTH2Ang->GetNbinsX();
    obserVariables[iVar4]->numbins = relEffTH2Ang->GetNbinsY();

    // std::vector< Variable*> massVars;
    // massVars.push_back(obserVariables[iVar1]);
    // massVars.push_back(obserVariables[iVar2]);

    effDatasetMasses = new BinnedDataSet(obserVariables,"efficiency Dataset Masses");
    effDatasetAngles = new BinnedDataSet(obserVariables,"efficiency Dataset Angles");
    effDataset = new BinnedDataSet(obserVariables,"efficiency Dataset Angles");

    //INITIALIZE TO ZERO

    for (int j = 0; j < effDatasetMasses->getNumBins(); ++j) {
      effDatasetMasses->setBinContent(j,0.0);
    }
    for (int j = 0; j < effDatasetAngles->getNumBins(); ++j) {
      effDatasetAngles->setBinContent(j,0.0);
    }

    // FILLING DATASET WITH HISTOGRAM
    for (int j = 0; j < effDatasetMasses->getNumBins(); ++j) {

      effDatasetMasses->setBinContent(j, relEffTH2Mass->GetBinContent(relEffTH2Mass->FindBin(effDatasetMasses->getBinCenter(massKPi,j),effDatasetMasses->getBinCenter(massPsiPi,j))));
      // if()
      // {
      //   std::cout<<"Histo content at massKpi : "<<effDatasetMasses->getBinCenter(massKPi,j)<<" and massPsiPi : " <<effDatasetMasses->getBinCenter(massPsiPi,j)<<" is = "<<relEffTH2Mass->GetBinContent(relEffTH2Mass->FindBin(effDatasetMasses->getBinCenter(massKPi,j),effDatasetMasses->getBinCenter(massPsiPi,j)))<<std::endl;
      //   std::cout<<"Binned dataset content : "<<effDatasetMasses->getBinContent(j)<<" at massKpi : "<<effDatasetMasses->getBinCenter(massKPi,j)<<" and massPsiPi : " <<effDatasetMasses->getBinCenter(massPsiPi,j)<<" Bin = "<<j<<std::endl;
      //
      // }
    }

    for (int j = 0; j < effDatasetAngles->getNumBins(); ++j) {

      effDatasetAngles->setBinContent(j,relEffTH2Ang->GetBinContent(relEffTH2Ang->FindBin(effDatasetAngles->getBinCenter(cosMuMu,j),effDatasetAngles->getBinCenter(phi,j))));
      // if((relEffTH2Ang->GetBinContent(relEffTH2Ang->FindBin(effDatasetAngles->getBinCenter(cosMuMu,j),effDatasetAngles->getBinCenter(phi,j)))!=0.0))
      // {
      //   std::cout<<"Histo content at massKpi : "<<effDatasetAngles->getBinCenter(cosMuMu,j)<<" and massPsiPi : " <<effDatasetAngles->getBinCenter(phi,j)<<" is = "<<relEffTH2Ang->GetBinContent(relEffTH2Ang->FindBin(effDatasetAngles->getBinCenter(cosMuMu,j),effDatasetAngles->getBinCenter(phi,j)))<<std::endl;
      //   std::cout<<"Binned dataset content : "<<effDatasetAngles->getBinContent(j)<<" at cosMuMu : "<<effDatasetAngles->getBinCenter(cosMuMu,j)<<" and phi : " <<effDatasetAngles->getBinCenter(phi,j)<<" Bin = "<<j<<std::endl;
      // }
    }

    for (int j = 0; j < effDataset->getNumBins(); ++j) {

      massKPi->value = effDataset->getBinCenter(massKPi,j);
      massPsiPi->value = effDataset->getBinCenter(massPsiPi,j);
      phi->value = effDataset->getBinCenter(phi,j);
      cosMuMu->value = effDataset->getBinCenter(cosMuMu,j);

      fptype anglEff = effDatasetAngles->getBinContent(effDatasetAngles->getBinNumber());
      fptype massEff = effDatasetMasses->getBinContent(effDatasetMasses->getBinNumber());

      effDatasetAngles->setBinContent(j,anglEff*massEff);
      // if(massEff!=0.0 && anglEff!=0.0)
      // {
      //   std::cout<<"MassKPi : "<<massKPi->value<<" - MassPsiPi : "<<massPsiPi->value<<" - Phi : "<<phi->value<<" - CosMuMu : "<<cosMuMu->value<<std::endl;
      //   std::cout<<"Ang efficiency "<<anglEff<<" and massPsiPi : " <<massEff<<" tot eff : "<<anglEff*massEff<<std::endl;
      //   std::cout<<"Histo content = "<<relEffTH2Mass->GetBinContent(relEffTH2Mass->FindBin(effDataset->getBinCenter(massKPi,j),effDatasetMasses->getBinCenter(massPsiPi,j)))<<std::endl;
      //   std::cout<<"Histo content = "<<relEffTH2Ang->GetBinContent(relEffTH2Ang->FindBin(effDataset->getBinCenter(cosMuMu,j),effDatasetMasses->getBinCenter(phi,j)))<<std::endl;
      //
      // }

    }


    efficiencyHistMasses = new FlatHistoPdf ("EfficiencyMassPdf",effDatasetMasses,obserVariables);
    efficiencyHistAngles = new FlatHistoPdf ("EfficiencyAnglesPdf",effDatasetAngles,obserVariables);
    effHist = new FlatHistoPdf ("EfficienciesPdf",effDataset,obserVariables);
    //efficiencyHistMasses = new BiDimHistoPdf ("EfficiencyPdf",effDatasetMasses,massVars,1);
    // UnbinnedDataSet plottingGridMasses(massVars);
    //
    //
    // for (int i = 0; i < massKPi->numbins; ++i)
    // {
    //     //pdfTestValues.push_back(0.0);
    //     massKPi->value = massKPi->lowerlimit + (massKPi->upperlimit - massKPi->lowerlimit)*(i + 0.5) / massKPi->numbins;
    //     for (int j = 0; j < massPsiPi->numbins; ++j)
    //     {
    //       massPsiPi->value = massPsiPi->lowerlimit + (massPsiPi->upperlimit - massPsiPi->lowerlimit)*(j + 0.5) / massPsiPi->numbins;
    //       plottingGridMasses.addEvent();
    //     }
    // }

    obserVariables[iVar1]->numbins = holdBinVar1;
    obserVariables[iVar2]->numbins = holdBinVar2;
    obserVariables[iVar1]->numbins = holdBinVar3;
    obserVariables[iVar2]->numbins = holdBinVar4;
    //return 0;
  }


  //PDFs
  GooPdf* totalPdf;

  vector<PdfBase*> pdfComponents;
  vector<Variable*> pdfYield;

  std::string p = "phasespace";

  //Variable* sFrac = new Variable("sFrac",0.5,0.,1.0);
  Variable* sFrac = new Variable("sFrac",0.7);
  //Variable* halfFrac = new Variable("halfFrac",0.25);

  GooPdf* matrix = new MatrixPdf("Kstars_signal", massKPi, cosMuMu, massPsiPi, phi,Masses,Gammas,Spins,as,bs,psi_nS,dRadB0,dRadKs);
  GooPdf* background;
  GooPdf* sumPdf;


  std::cout<<"Initliasing p.d.f.s components"<<std::endl;
  if (bkgPhaseSpace)
  {
    std::cout<<"- Bakground phase-space p.d.f."<<std::endl;
    background = new ThreeBodiesPsiPiK("phasespace",massKPi,cosMuMu,massPsiPi,phi,&mBd,&mPion,&mKaon,&mMuMu);
    sumPdf     = new AddPdf("Kstars_signal + PhaseSpace", sFrac, matrix,background);
    if (effPdfProd)
    {
      std::cout<<"- Efficiency map p.d.f."<<std::endl;
      pdfComponents.push_back(sumPdf);
      pdfComponents.push_back(effHist);
      // pdfComponents.push_back(efficiencyHistMasses);
      // pdfComponents.push_back(efficiencyHistAngles);
      totalPdf = new ProdPdf("(Kstars_signal + phaseSpace) * efficiency",pdfComponents);

    }
    else
      totalPdf = sumPdf;
  }
  else {
    if (bkgPhaseSpaceMap) {
      std::cout<<"- Background map p.d.f."<<std::endl;

      int iVar1 = 0, iVar2 = 1, iVar3 = 2, iVar4 = 3;
      int holdBinVar1, holdBinVar2,holdBinVar4,holdBinVar3;

      // path = "./datafiles/";
      //TString bkgName = "Data_JPsi_2p0Sig_6p0to9p0SB.root";
      TString bkgName = "TMVApp_data_withBDTCutAt0p00_JPsi_2p0Sig_6p0to9p0SB.root";
      TFile *bkgFile = TFile::Open(path+bkgName);

      TString bkgNameMass = "psi2SPi_vs_KPi_masses_sbs_BDT";
      TString bkgNameAng = "planesAngle_vs_cos_psi2S_helicityAngle_sbs_BDT";

      bkgTH2Mass = (TH2F*)bkgFile->Get(bkgNameMass) ;
      bkgTH2Ang = (TH2F*)bkgFile->Get(bkgNameAng) ;

      if (!(bkgTH2Mass)) {
        std::cout<<"Efficiency TH2 named \'"<<bkgNameMass<<"\' NOT FOUND in found in TFile \'" <<bkgFile->GetName() <<"\'.\nReturning."<<std::endl;
        return -1;
      }
      if (!(bkgTH2Ang)) {
        std::cout<<"Efficiency TH2 named \'"<<bkgNameAng<<"\' NOT FOUND in found in TFile \'" <<bkgFile->GetName() <<"\'.\nReturning."<<std::endl;
        return -1;
      }

      bkgTH2Mass->Scale(1/bkgTH2Mass->GetEntries());
      bkgTH2Ang->Scale(1/bkgTH2Ang->GetEntries());

      bkgMKPi = bkgTH2Mass->ProjectionX(); bkgMPsiPi = bkgTH2Mass->ProjectionY();
      bkgCMuMu = bkgTH2Ang->ProjectionX(); bkgPhi = bkgTH2Ang->ProjectionY();

      std::cout<<"Masses Sidebands TH2 read with bin massKPi = " <<bkgTH2Mass->GetNbinsX() <<" and bin massPsiPi = " <<bkgTH2Mass->GetNbinsY() <<std::endl;
      std::cout<<"Angles Sidebands TH2 read with bin cosMuMu = " <<bkgTH2Ang->GetNbinsX() <<" and bin phi = " <<bkgTH2Ang->GetNbinsY() <<std::endl;

      holdBinVar1 = obserVariables[iVar1]->numbins;
      holdBinVar2 = obserVariables[iVar2]->numbins;
      holdBinVar3 = obserVariables[iVar3]->numbins;
      holdBinVar4 = obserVariables[iVar4]->numbins;

      obserVariables[iVar1]->numbins = bkgTH2Mass->GetNbinsX();
      obserVariables[iVar2]->numbins = bkgTH2Mass->GetNbinsY();
      obserVariables[iVar3]->numbins = bkgTH2Ang->GetNbinsX();
      obserVariables[iVar4]->numbins = bkgTH2Ang->GetNbinsY();

      bkgDatasetMasses = new BinnedDataSet(obserVariables,"bkg Dataset Masses");
      bkgDatasetAngles = new BinnedDataSet(obserVariables,"bkg Dataset Angles");
      bkgDataset = new BinnedDataSet(obserVariables,"bkg Dataset");
      //INITIALIZE TO ZERO

      for (int j = 0; j < bkgDatasetMasses->getNumBins(); ++j)
        bkgDatasetMasses->setBinContent(j,0.0);
      for (int j = 0; j < bkgDatasetAngles->getNumBins(); ++j)
        bkgDatasetAngles->setBinContent(j,0.0);
      for (int j = 0; j < bkgDataset->getNumBins(); ++j)
        bkgDataset->setBinContent(j,0.0);


      // FILLING DATASET WITH HISTOGRAM
      for (int j = 0; j < bkgDatasetMasses->getNumBins(); ++j) {

        bkgDatasetMasses->setBinContent(j, bkgTH2Mass->GetBinContent(bkgTH2Mass->FindBin(bkgDatasetMasses->getBinCenter(massKPi,j),bkgDatasetMasses->getBinCenter(massPsiPi,j))));
        // if((bkgTH2Mass->GetBinContent(bkgTH2Mass->FindBin(bkgDatasetMasses->getBinCenter(massKPi,j),bkgDatasetMasses->getBinCenter(massPsiPi,j)))!=0.))
        // {
        //   std::cout<<"Histo content at massKpi : "<<bkgDatasetMasses->getBinCenter(massKPi,j)<<" and massPsiPi : " <<bkgDatasetMasses->getBinCenter(massPsiPi,j)<<" is = "<<bkgTH2Mass->GetBinContent(bkgTH2Mass->FindBin(bkgDatasetMasses->getBinCenter(massKPi,j),bkgDatasetMasses->getBinCenter(massPsiPi,j)))<<std::endl;
        //   std::cout<<"Binned dataset content : "<<bkgDatasetMasses->getBinContent(j)<<" at massKpi : "<<bkgDatasetMasses->getBinCenter(massKPi,j)<<" and massPsiPi : " <<bkgDatasetMasses->getBinCenter(massPsiPi,j)<<" Bin = "<<j<<std::endl;
        // }
      }

      for (int j = 0; j < bkgDatasetAngles->getNumBins(); ++j) {

        bkgDatasetAngles->setBinContent(j, bkgTH2Ang->GetBinContent(bkgTH2Ang->FindBin(bkgDatasetAngles->getBinCenter(cosMuMu,j),bkgDatasetAngles->getBinCenter(phi,j))));
        //if ((bkgTH2Ang->GetBinContent(bkgTH2Ang->FindBin(bkgDatasetAngles->getBinCenter(cosMuMu,j),bkgDatasetAngles->getBinCenter(phi,j))))!=0.) {
        // std::cout<<"Histo content at phi : "<<bkgDatasetMasses->getBinCenter(phi,j)<<" and cosMuMu : " <<bkgDatasetMasses->getBinCenter(cosMuMu,j)<<" is = "<<bkgTH2Ang->GetBinContent(bkgTH2Ang->FindBin(bkgDatasetMasses->getBinCenter(phi,j),bkgDatasetMasses->getBinCenter(cosMuMu,j)))<<std::endl;
        // std::cout<<"Binned dataset content : "<<bkgDatasetMasses->getBinContent(j)<<" at phi : "<<bkgDatasetMasses->getBinCenter(phi,j)<<" and cosMuMu : " <<bkgDatasetMasses->getBinCenter(cosMuMu,j)<<" Bin = "<<j<<std::endl;
	//}
      }

      for (int j = 0; j < bkgDataset->getNumBins(); ++j) {
        fptype anglesContent = bkgTH2Ang->GetBinContent(bkgTH2Ang->FindBin(bkgDataset->getBinCenter(phi,j), bkgDataset->getBinCenter(cosMuMu,j)));
        fptype massesContent = bkgTH2Mass->GetBinContent(bkgTH2Mass->FindBin(bkgDataset->getBinCenter(massPsiPi,j), bkgDataset->getBinCenter(massKPi,j)));

        fptype content = anglesContent * massesContent;

        bkgDataset->setBinContent(j,content);
      }

      //int noOfEntries = bkgTH2Mass->GetEntries() + bkgTH2Ang->GetEntries();
      //std::cout<<"Dataset events : " <<bkgDataset->getNumEvents()<<" histograms : " <<noOfEntries <<std::endl;
      //std::cout<<"Mass histo : " <<bkgTH2Mass->GetEntries()<<" Angles histo : " <<bkgTH2Ang->GetEntries() <<std::endl;

      bkgHistMasses = new FlatHistoPdf ("bkgHistMasses",bkgDatasetMasses,obserVariables);
      bkgHistAngles = new FlatHistoPdf ("bkgHistAngles",bkgDatasetAngles,obserVariables);

      //efficiencyHistMasses = new BiDimHistoPdf ("EfficiencyPdf",bkgDatasetMasses,massVars,1);
      // UnbinnedDataSet plottingGridMasses(massVars);
      //
      //
      // for (int i = 0; i < massKPi->numbins; ++i)
      // {
      //     //pdfTestValues.push_back(0.0);
      //     massKPi->value = massKPi->lowerlimit + (massKPi->upperlimit - massKPi->lowerlimit)*(i + 0.5) / massKPi->numbins;
      //     for (int j = 0; j < massPsiPi->numbins; ++j)
      //     {
      //       massPsiPi->value = massPsiPi->lowerlimit + (massPsiPi->upperlimit - massPsiPi->lowerlimit)*(j + 0.5) / massPsiPi->numbins;
      //       plottingGridMasses.addEvent();
      //     }
      // }

      obserVariables[iVar1]->numbins = holdBinVar1;
      obserVariables[iVar2]->numbins = holdBinVar2;
      obserVariables[iVar1]->numbins = holdBinVar3;
      obserVariables[iVar2]->numbins = holdBinVar4;

      std::vector<Variable*> weights;

      pdfComponents.push_back(bkgHistMasses);
      pdfComponents.push_back(bkgHistAngles);

      //weights.push_back(sFrac);
      //weights.push_back(halfFrac);

      //background = new AddPdf("Background P.d.f Map",halfFrac,bkgHistAngles,bkgHistMasses);
      //background = new FlatHistoPdf ("bkgHistAngles",bkgDataset,obserVariables);
      background = new ProdPdf("background",pdfComponents);
      sumPdf = new AddPdf("Kstars_signal + combinatorial", sFrac, matrix, background);
      //sumPdf = new AddPdf("Kstars_signal + PhaseSpace",weights,pdfComponents);

      pdfComponents.clear();

      if (effPdfProd) {
        std::cout<<"- Efficiency p.d.f. "<<std::endl;
        pdfComponents.push_back(sumPdf);
        pdfComponents.push_back(efficiencyHistMasses);
        pdfComponents.push_back(efficiencyHistAngles);

        totalPdf = new ProdPdf("(Kstars_signal + combinatorial) * efficiency",pdfComponents);
      }
      else
        totalPdf = sumPdf;
    } // if (bkgPhaseSpaceMap)
    else {
      if(effPdfProd) {
	pdfComponents.push_back(matrix);
	pdfComponents.push_back(efficiencyHistMasses);

	totalPdf = new ProdPdf("Kstars_signal * efficiency",pdfComponents);
	//totalPdf = matrix;
      }
      else
	totalPdf = matrix;
    }
  }

  pdfComponents.clear();
  pdfYield.clear();

  obserVariables[0]->numbins = bin1;
  obserVariables[1]->numbins = bin2;

  totalPdf->setData(&dataset);
  //total->setData(&dataset);

  cout <<"\n- Fitting ..." <<endl;
  //FitManager fitter(total);
  // FitManager* fitter = 0;
  // if (!hesse)
  //   fitter = new FitManager(totalPdf);
  // else
  //   fitter = new FitManager(totalPdf,hesse);

  FitManager* fitter = new FitManager(totalPdf);

  totalPdf->setFitControl(new UnbinnedNllFit());


  gettimeofday(&startTime, NULL);
  startC = times(&startProc);
  //
  //fitter->fitOrdered(algos);
  fitter->fit();
  fitter->getMinuitValues();

  std::vector<fptype> originalAs;
  std::vector<fptype> originalBs;

  for (int i = 0; i < nHelAmps; i++) {
    originalAs.push_back(as[i]->value);
    originalBs.push_back(bs[i]->value);
  }

  //
  stopC = times(&stopProc);
  gettimeofday(&stopTime, NULL);
  fptype fitClocks = (stopC - startC)*10000.;

  // Bring phases within [-TMath::Pi,+TMath::Pi]
  fptype period = 2*TMATH_PI;
  for (int i = 0; i < nHelAmps; i++) {
    while (fabs(bs[i]->value) > TMATH_PI)
      bs[i]->value += bs[i]->value > 0 ? -period : +period ;
  }
  gettimeofday(&startTime, NULL);
  startC = times(&startProc);
  //
  UnbinnedDataSet plottingGridData(obserVariables);

  std::vector<UnbinnedDataSet> compData;
  std::vector<std::vector<fptype> > pdfTotalValues;
  std::vector<std::vector<fptype> > pdfTotalSigValues;
  std::vector<std::vector<fptype> > pdfTotalBkgValues;
  /*std::vector<std::vector<std::vector<fptype> >  pdfCompValues;*/
  std::vector<std::vector<fptype> > pdfCompValues;

  for (int id = 0; id < nKstars; ++id) {
    compData.push_back( UnbinnedDataSet(obserVariables) );
  }

  std::vector<fptype> fractions;

  std::vector<fptype> compEvents;

  std::vector<fptype> mkpTotalProjection, mkpTotalSigProjection, mkpTotalBkgProjection;
  std::vector<fptype> cosMuMuTotalProjection, cosMuMuTotalSigProjection, cosMuMuTotalBkgProjection;
  std::vector<fptype> massPsiPiTotalProjection, massPsiPiTotalSigProjection, massPsiPiTotalBkgProjection;
  std::vector<fptype> phiTotalProjection, phiTotalSigProjection, phiTotalBkgProjection;

  massKPi->numbins = plottingfine1;
  cosMuMu->numbins = plottingfine2;
  massPsiPi->numbins = plottingfine3;
  phi->numbins = plottingfine4;

  fptype pointsMKPiXTot[massKPi->numbins];
  fptype pointsMKPiYTot[massKPi->numbins],pointsMKPiYTotSig[massKPi->numbins],pointsMKPiYTotBkg[massKPi->numbins];

  fptype pointsCosMuMuXTot[cosMuMu->numbins];
  fptype pointsCosMuMuYTot[cosMuMu->numbins];

  fptype pointsmassPsiPiXTot[massPsiPi->numbins];
  fptype pointsmassPsiPiYTot[massPsiPi->numbins];

  fptype pointsPhiXTot[phi->numbins];
  fptype pointsPhiYTot[phi->numbins];

  //Total pdf projection histos
  TH1F projMKPiHisto("projMKPiHisto", "projMKPiHisto",massKPi->numbins, massKPi->lowerlimit, massKPi->upperlimit);
  TH1F projCosMuMuHisto("projCosMuMuHisto", "projCosMuMuHisto",cosMuMu->numbins, cosMuMu->lowerlimit, cosMuMu->upperlimit);
  TH1F projmassPsiPiHisto("projmassPsiPiHisto", "projmassPsiPiHisto",massPsiPi->numbins, massPsiPi->lowerlimit, massPsiPi->upperlimit);
  TH1F projPhiHisto("projPhiHisto", "projPhiHisto",phi->numbins, phi->lowerlimit, phi->upperlimit);
  //Signal pdf projection histos
  TH1F projMKPiHistoSig("projMKPiHistoSignal", "projMKPiHistoSignal",massKPi->numbins, massKPi->lowerlimit, massKPi->upperlimit);
  TH1F projCosMuMuHistoSig("projCosMuMuHistoSignal", "projCosMuMuHistoSignal",cosMuMu->numbins, cosMuMu->lowerlimit, cosMuMu->upperlimit);
  TH1F projmassPsiPiHistoSig("projmassPsiPiHistoSignal", "projmassPsiPiHistoSignal",massPsiPi->numbins, massPsiPi->lowerlimit, massPsiPi->upperlimit);
  TH1F projPhiHistoSig("projPhiHistoSignal", "projPhiHistoSignal",phi->numbins, phi->lowerlimit, phi->upperlimit);
  //Background pdf projection histos
  TH1F projMKPiHistoBkg("projMKPiHistoBkg", "projMKPiHistoBkg",massKPi->numbins, massKPi->lowerlimit, massKPi->upperlimit);
  TH1F projCosMuMuHistoBkg("projCosMuMuHistoBkg", "projCosMuMuHistoBkg",cosMuMu->numbins, cosMuMu->lowerlimit, cosMuMu->upperlimit);
  TH1F projmassPsiPiHistoBkg("projmassPsiPiHistoBkg", "projmassPsiPiHistoBkg",massPsiPi->numbins, massPsiPi->lowerlimit, massPsiPi->upperlimit);
  TH1F projPhiHistoBkg("projPhiHistoBkg", "projPhiHistoBkg",phi->numbins, phi->lowerlimit, phi->upperlimit);


  for (int i = 0; i < massKPi->numbins; ++i)
    {
      mkpTotalProjection.push_back(0.0);
      mkpTotalSigProjection.push_back(0.0);
      mkpTotalBkgProjection.push_back(0.0);
    }

  for (int i = 0; i < cosMuMu->numbins; ++i)
    {
      cosMuMuTotalProjection.push_back(0.0);
      cosMuMuTotalSigProjection.push_back(0.0);
      cosMuMuTotalBkgProjection.push_back(0.0);
    }

  for (int i = 0; i < massPsiPi->numbins; ++i)
    {
      massPsiPiTotalProjection.push_back(0.0);
      massPsiPiTotalSigProjection.push_back(0.0);
      massPsiPiTotalBkgProjection.push_back(0.0);
    }

  for (int i = 0; i < phi->numbins; ++i)
    {
      phiTotalProjection.push_back(0.0);
      phiTotalSigProjection.push_back(0.0);
      phiTotalBkgProjection.push_back(0.0);
    }

  fptype sum = 0.0;
  fptype sumSig = 0.0;
  fptype sumBkg = 0.0;

  std::cout <<"\n- Starting plotting cycle" ;
  std::cout <<"\n- Plotting generated dataset" <<std::endl;
  for (int k = 0; k < phi->numbins; ++k) {
    phi->value = phi->lowerlimit + (phi->upperlimit - phi->lowerlimit)*(k + 0.5) / phi->numbins;
    //std::cout <<"Phi : " << k <<std::endl;
    for (int j = 0; j < cosMuMu->numbins; ++j) {
      cosMuMu->value = cosMuMu->lowerlimit + (cosMuMu->upperlimit - cosMuMu->lowerlimit)*(j + 0.5) / cosMuMu->numbins;
      //std::cout <<"CosMu : " << j <<std::endl;
      for (int a = 0; a < massPsiPi->numbins; ++a) {
	massPsiPi->value = massPsiPi->lowerlimit + (massPsiPi->upperlimit - massPsiPi->lowerlimit)*(a + 0.5) / massPsiPi->numbins;
	//std::cout <<"CosK : " << a <<std::endl;
	for (int i = 0; i < massKPi->numbins; ++i) {
	  //std::vector<std::vector<fptype> > tempValues;
	  //UnbinnedDataSet tempData(obserVariables);
	  massKPi->value = massKPi->lowerlimit + (massKPi->upperlimit - massKPi->lowerlimit)*(i + 0.5) / massKPi->numbins;
	  //std::cout <<"MKP : " << i <<std::endl;

	  /*tempData.addEvent();
	    matrix->setData(&tempData);
	    matrix->getCompProbsAtDataPoints(tempValues);

	    std::cout <<massKPi->value<<" ";
	    std::cout <<cosMuMu->value<<" ";
	    std::cout <<massPsiPi->value<<" ";
	    std::cout <<phi->value<<" ";
	    std::cout <<tempValues[0][0]<<std::endl;*/

	  //mkpTotalProjection[i]+=tempValues[0][0];
	  //sum +=tempValues[0][0];

	  plottingGridData.addEvent();
	  /*for (size_t ii = 0; ii < compData.size(); ++ii) {
	    compData[ii].addEvent();
	    }*/
	}
      }
    }
  }
  //
  stopC = times(&stopProc);
  gettimeofday(&stopTime, NULL);
  fptype dataSetClocks = (stopC - startC)*10000.;

  ////////////////////////////////////////////////////////////////////////////////
  ///// TOTAL PDF PLOT
  ////////////////////////////////////////////////////////////////////////////////

  Float_t xMax = 0.95, yMax = 0.9;
  Float_t legLeft = 0.6, legWidth = 0.15;
  TLegend *legPlot = new TLegend(legLeft, 0.6, legLeft+legWidth, yMax); // 0.6 will be replaced later
  TPaveText *fitStat = new TPaveText(legPlot->GetX2(), 0.4, xMax, yMax, "NDC");

  std::cout <<"\n- Evaluating the total p.d.f." <<std::endl;
  totalPdf->setData(&plottingGridData);

  gettimeofday(&startTime, NULL);
  startC = times(&startProc);
  //
  if(bkgPhaseSpaceMap && effPdfProd)
  {
    sumPdf->setData(&plottingGridData);
    sumPdf->getCompProbsAtDataPoints(pdfTotalValues);
  }
  else
    totalPdf->getCompProbsAtDataPoints(pdfTotalValues);

  int indexComponents = 0;

  // if (effPdfProd) {
  //   indexComponents++;
  // }
  std::cout <<" Vector size : " <<pdfTotalValues.size()<<std::endl;
  //std::cout <<" Vector proj : " <<pdfTotalValues[0].size()/massKPi->numbins<<std::endl;
  for (int k = 0; k < pdfTotalValues[0+indexComponents].size(); k++) {
    //std::cout <<mkpTotalProjection[k]*events/sum<<std::endl;
    sum += pdfTotalValues[0+indexComponents][k];
      if(bkgPhaseSpace && !effPdfProd)
      {
        sumSig += pdfTotalValues[1][k];
        sumBkg += pdfTotalValues[2][k];
      }
  }
  //
  stopC = times(&stopProc);
  gettimeofday(&stopTime, NULL);
  fptype sumClocks = (stopC - startC)*10000.;

  gettimeofday(&startTime, NULL);
  startC = times(&startProc);
  //
  Float_t sigFrac = sFrac->value;
  Float_t bkgFrac = 1 - sigFrac;
  std::cout <<"\n[ Total Pdf sum : " <<sum<<" ] " <<std::endl;
  for (int k = 0; k<pdfTotalValues[0].size(); ++k) {
    //std::cout <<mkpTotalProjection[k]*events/sum<<std::endl;
    pdfTotalValues[0+indexComponents][k] /= sum;
    pdfTotalValues[0+indexComponents][k] *= events;
    if(bkgPhaseSpace && !effPdfProd)
    {
      pdfTotalValues[1][k] /= sumSig;
      pdfTotalValues[1][k] *= (events*sigFrac);

      pdfTotalValues[2][k] /= sumBkg;
      pdfTotalValues[2][k] *= (events*bkgFrac);
    }
  }
  //
  stopC = times(&stopProc);
  gettimeofday(&stopTime, NULL);
  startC = times(&startProc);
  fptype normClocks = (stopC - startC)*10000.;

  //////////////////////////////////////////////////////////////////////
  //PROJECTING PDF ON THE FOUR VARIABLES (mkpi,phi,cosMuMu,cosK)
  //////////////////////////////////////////////////////////////////////
  //   Pdf evaluation vector (pdfTotalValues) structure :
  //
  //   es.
  //   b1 = 4  mKPi
  //   b2 = 5  massPsiPi
  //   b3 = 3  cosMuMu
  //   b4 = 3  phi
  //   ==================================================
  //   array ind   coordinates where the pdf is evaluated
  //   ==================================================
  //   0                phi0 cosMu0 cosk0 mkp0
  //   1                phi0 cosMu0 cosk0 mkp1
  //   2                phi0 cosMu0 cosk0 mkp2
  //   3                phi0 cosMu0 cosk0 mkp3
  //   4(b1)            phi0 cosMu0 cosk1 mkp0
  //   5                phi0 cosMu0 cosk1 mkp1
  //   . . .
  //   19               phi0 cosMu0 cosk4 mkp3
  //   20(b1*b2)        phi0 cosMu1 cosk0 mkp0
  //   . . .
  //   59               phi0 cosMu2 cosk4 mkp3
  //   60(b1*b2*b3)     phi1 cosMu0 cosk0 mkp0
  //   . . .            phi0 cosMu0 cosk0 mkp3
  //   179              phi2 cosMu2 cosk4 mkp3

  int notMPKBins = pdfTotalValues[0].size()/massKPi->numbins;
  int notCosMuMuBins = pdfTotalValues[0].size()/cosMuMu->numbins;
  int notmassPsiPiBins = pdfTotalValues[0].size()/massPsiPi->numbins;
  int notPhiBins = pdfTotalValues[0].size()/phi->numbins;

  //Mass K Pi
  for (int j = 0; j < massKPi->numbins; ++j) {
    for (int i = 0; i < notMPKBins; ++i) {
      mkpTotalProjection[j] += pdfTotalValues[0+indexComponents][j  +  i * massKPi->numbins];
      if(bkgPhaseSpace && !effPdfProd)
      {
        mkpTotalSigProjection[j] += pdfTotalValues[1][j  +  i * massKPi->numbins];
        mkpTotalBkgProjection[j] += pdfTotalValues[2][j  +  i * massKPi->numbins];
      }
    }
  }

  //Cos Mu Mu
  for (int j = 0; j < massPsiPi->numbins; ++j) {
    for (int k = 0; k < phi->numbins * cosMuMu->numbins; ++k) {
      for (int i = 0; i < massKPi->numbins; ++i) {
        massPsiPiTotalProjection[j] += pdfTotalValues[0+indexComponents][i  +  k * massKPi->numbins * massPsiPi->numbins  +  j * massKPi->numbins];
        // if(bkgPhaseSpace)
        // {
        //   massPsiPiTotalSigProjection[j] += pdfTotalValues[1][i  +  k * massKPi->numbins * massPsiPi->numbins  +  j * massKPi->numbins];
        //   massPsiPiTotalBkgProjection[j] += pdfTotalValues[2][i  +  k * massKPi->numbins * massPsiPi->numbins  +  j * massKPi->numbins];
        // }
      }
    }
  }

  // //Cos Mu Mu
  // for (int j = 0; j < cosMuMu->numbins; ++j) {
  //  for (int k = 0; k < phi->numbins*cosMuMu->numbins; ++k) {
  //    for (int i = 0; i < massKPi->numbins; ++i) {
  //      cosMuMuTotalProjection[j] += pdfTotalValues[0][i+k*massKPi->numbins*massPsiPi->numbins+j*massKPi->numbins];
  //    }
  //  }
  // }

  // //Cos K Star
  // for (int j = 0; j < massPsiPi->numbins; ++j) {
  //  for (int k = 0; k < phi->numbins; ++k) {
  //    for (int i = 0; i < massKPi->numbins*cosMuMu->numbins; ++i) {
  //      massPsiPiTotalProjection[j] += pdfTotalValues[0][i+j*massKPi->numbins*massPsiPi->numbins+k*massKPi->numbins*cosMuMu->numbins*massPsiPi->numbins];
  //    }
  //  }
  // }

  //Cos K Star
  for (int j = 0; j < cosMuMu->numbins; ++j) {
    for (int k = 0; k < phi->numbins; ++k) {
      for (int i = 0; i < massKPi->numbins*cosMuMu->numbins; ++i) {
        cosMuMuTotalProjection[j] += pdfTotalValues[0+indexComponents][i  +  j * massKPi->numbins * massPsiPi->numbins  +  k * massKPi->numbins * cosMuMu->numbins * massPsiPi->numbins];
        // if(bkgPhaseSpace)
        // {
        //   cosMuMuTotalSigProjection[j] += pdfTotalValues[1][i  +  j * massKPi->numbins * massPsiPi->numbins  +  k * massKPi->numbins * cosMuMu->numbins * massPsiPi->numbins];
        //   cosMuMuTotalBkgProjection[j] += pdfTotalValues[2][i  +  j * massKPi->numbins * massPsiPi->numbins  +  k * massKPi->numbins * cosMuMu->numbins * massPsiPi->numbins];
        // }
      }
    }
  }

  //Phi
  for (int j = 0; j < phi->numbins; ++j) {
    for (int k = 0; k < massPsiPi->numbins * massKPi->numbins * cosMuMu->numbins; ++k) {
        phiTotalProjection[j] += pdfTotalValues[0+indexComponents][k  +  j * massKPi->numbins * cosMuMu->numbins * massPsiPi->numbins];
        // if(bkgPhaseSpace)
        // {
        //   phiTotalSigProjection[j] += pdfTotalValues[1][k  +  j * massKPi->numbins * cosMuMu->numbins * massPsiPi->numbins];
        //   phiTotalBkgProjection[j] += pdfTotalValues[2][k  +  j * massKPi->numbins * cosMuMu->numbins * massPsiPi->numbins];
        // }
    }
  }

  //////////////////////////////////////////////////////////////////////
  //Timing
  stopC = times(&stopProc);
  gettimeofday(&stopTime, NULL);

  fptype projClocks = (stopC - startC)*10000.;

  //////////////////////////////////////////////////////////////////////
  //Fillling projection histograms

  for (int j = 0; j < massKPi->numbins; ++j) {
    projMKPiHisto.SetBinContent(j+1,mkpTotalProjection[j]);
    projMKPiHistoSig.SetBinContent(j+1,mkpTotalSigProjection[j]);
    projMKPiHistoBkg.SetBinContent(j+1,mkpTotalBkgProjection[j]);
    //std::cout <<" Bin " <<j<<" center = " <<projMKPiHisto.GetBinCenter(j+1)<<" : " <<mkpTotalProjection[j]<<std::endl;
  }

  projMKPiHisto.Scale(ratioMKPi);
  projMKPiHistoSig.Scale(ratioMKPi);
  projMKPiHistoBkg.Scale(ratioMKPi);

  for (int j = 0; j < cosMuMu->numbins; ++j) {
    projCosMuMuHisto.SetBinContent(j+1,cosMuMuTotalProjection[j]);
    // projCosMuMuHistoSig.SetBinContent(j+1,cosMuMuTotalSigProjection[j]);
    // projCosMuMuHistoBkg.SetBinContent(j+1,cosMuMuTotalBkgProjection[j]);
    //std::cout <<" Bin " <<j<<" center = " <<projMKPiHisto.GetBinCenter(j+1)<<" : " <<mkpTotalProjection[j]<<std::endl;
  }

  projCosMuMuHisto.Scale(ratioCosMuMu);
  // projCosMuMuHistoSig.Scale(ratioCosMuMu);
  // projCosMuMuHistoBkg.Scale(ratioCosMuMu);

  for (int j = 0; j < massPsiPi->numbins; ++j) {
    projmassPsiPiHisto.SetBinContent(j+1,massPsiPiTotalProjection[j]);
    // projmassPsiPiHistoSig.SetBinContent(j+1,massPsiPiTotalSigProjection[j]);
    // projmassPsiPiHistoBkg.SetBinContent(j+1,massPsiPiTotalBkgProjection[j]);
    //std::cout <<" Bin " <<j<<" center = " <<projMKPiHisto.GetBinCenter(j+1)<<" : " <<mkpTotalProjection[j]<<std::endl;
  }

  projmassPsiPiHisto.Scale(ratioMassPsiPi);
  // projmassPsiPiHistoSig.Scale(ratioMassPsiPi);
  // projmassPsiPiHistoBkg.Scale(ratioMassPsiPi);

  for (int j = 0; j < phi->numbins; ++j) {
    projPhiHisto.SetBinContent(j+1,phiTotalProjection[j]);
    // projPhiHistoSig.SetBinContent(j+1,phiTotalSigProjection[j]);
    // projPhiHistoBkg.SetBinContent(j+1,phiTotalBkgProjection[j]);
    //std::cout <<" Bin " <<j<<" center = " <<projMKPiHisto.GetBinCenter(j+1)<<" : " <<mkpTotalProjection[j]<<std::endl;
  }

  projPhiHisto.Scale(ratioPhi);
  // projPhiHistoSig.Scale(ratioPhi);
  // projPhiHistoBkg.Scale(ratioPhi);

  //////////////////////////////////////////////////////////////////////
  //Fillling projection histograms and TGraphs

  for (int j = 0; j < massKPi->numbins; ++j) {
    pointsMKPiXTot[j] = projMKPiHisto.GetBinCenter(j+1);
    pointsMKPiYTot[j] = projMKPiHisto.GetBinContent(j+1);
    //std::cout <<" Bin " <<j<<" center = " <<projMKPiHisto.GetBinCenter(j+1)<<" : " <<mkpTotalProjection[j]<<std::endl;
  }

  for (int j = 0; j < cosMuMu->numbins; ++j) {
    pointsCosMuMuXTot[j] = projCosMuMuHisto.GetBinCenter(j+1);
    pointsCosMuMuYTot[j] = projCosMuMuHisto.GetBinContent(j+1);
    //std::cout <<" Bin " <<j<<" center = " <<projMKPiHisto.GetBinCenter(j+1)<<" : " <<mkpTotalProjection[j]<<std::endl;
  }

  for (int j = 0; j < massPsiPi->numbins; ++j) {
    pointsmassPsiPiXTot[j] = projmassPsiPiHisto.GetBinCenter(j+1);
    pointsmassPsiPiYTot[j] = projmassPsiPiHisto.GetBinContent(j+1);
    //std::cout <<" Bin " <<j<<" center = " <<projMKPiHisto.GetBinCenter(j+1)<<" : " <<mkpTotalProjection[j]<<std::endl;
  }

  for (int j = 0; j < phi->numbins; ++j) {
    pointsPhiXTot[j] = projPhiHisto.GetBinCenter(j+1);
    pointsPhiYTot[j] = projPhiHisto.GetBinContent(j+1);
    //std::cout <<" Bin " <<j<<" center = " <<projMKPiHisto.GetBinCenter(j+1)<<" : " <<mkpTotalProjection[j]<<std::endl;
  }

  //projPhiHisto.Scale(ratioPhi);

  //////////////////////////////////////////////////////////////////////
  //Setting Graphs & MultiGraphs

  TMultiGraph* multiGraphMKPi = new TMultiGraph(massKPi_name+"_MultiGraph", TString::Format("%s;%s",massKPiHisto.GetTitle(),(massKPiHisto.GetXaxis())->GetTitle()));
  TMultiGraph* multiGraphCosMuMu = new TMultiGraph(cosMuMu_name+"_MultiGraph", TString::Format("%s;%s",cosMuMuHisto.GetTitle(),cosMuMuHisto.GetXaxis()->GetTitle()));
  TMultiGraph* multiGraphmassPsiPi = new TMultiGraph(massPsiPi_name+"_MultiGraph", TString::Format("%s;%s",massPsiPiHisto.GetTitle(),massPsiPiHisto.GetXaxis()->GetTitle()));
  TMultiGraph* multiGraphPhi = new TMultiGraph(phi_name+"_MultiGraph", TString::Format("%s;%s",phiHisto.GetTitle(),phiHisto.GetXaxis()->GetTitle()));

  TGraph signalTotalPlotMKPi(massKPi->numbins,pointsMKPiXTot,pointsMKPiYTot);
  TGraph signalTotalSigPlotMKPi(massKPi->numbins,pointsMKPiXTot,pointsMKPiYTotSig);
  TGraph signalTotalBkgPlotMKPi(massKPi->numbins,pointsMKPiXTot,pointsMKPiYTotBkg);

  TGraph signalTotalPlotCosMuMu(cosMuMu->numbins,pointsCosMuMuXTot,pointsCosMuMuYTot);
  // TGraph signalTotalSigPlotCosMuMu(cosMuMu->numbins,pointsCosMuMuXTot,pointsCosMuMuYTotSig);
  // TGraph signalTotalBkgPlotCosMuMu(cosMuMu->numbins,pointsCosMuMuXTot,pointsCosMuMuYTotBkg);

  TGraph signalTotalPlotmassPsiPi(massPsiPi->numbins,pointsmassPsiPiXTot,pointsmassPsiPiYTot);
  // TGraph signalTotalSigPlotmassPsiPi(massPsiPi->numbins,pointsmassPsiPiXTot,pointsmassPsiPiYTotSig);
  // TGraph signalTotalBkgPlotmassPsiPi(massPsiPi->numbins,pointsmassPsiPiXTot,pointsmassPsiPiYTotBkg);

  TGraph signalTotalPlotPhi(phi->numbins,pointsPhiXTot,pointsPhiYTot);
  // TGraph signalTotalSigPlotPhi(phi->numbins,pointsPhiXTot,pointsPhiYTotSig);
  // TGraph signalTotalBkgPlotPhi(phi->numbins,pointsPhiXTot,pointsPhiYTotBkg);

  signalTotalPlotMKPi.SetLineColor(kRed); signalTotalPlotMKPi.SetLineWidth(2);
  signalTotalSigPlotMKPi.SetLineColor(kRed); signalTotalSigPlotMKPi.SetLineWidth(2); signalTotalSigPlotMKPi.SetLineStyle(kDashDotted);
  signalTotalBkgPlotMKPi.SetLineColor(kRed); signalTotalBkgPlotMKPi.SetLineWidth(2); signalTotalBkgPlotMKPi.SetLineStyle(kDashed);

  signalTotalPlotCosMuMu.SetLineColor(kRed); signalTotalPlotCosMuMu.SetLineWidth(2);
  // signalTotalSigPlotCosMuMu.SetLineColor(kRed); signalTotalSigPlotCosMuMu.SetLineWidth(2); signalTotalSigPlotCosMuMu.SetLineStyle(kDashDotted);
  // signalTotalBkgPlotCosMuMu.SetLineColor(kRed); signalTotalBkgPlotCosMuMu.SetLineWidth(2); signalTotalSigPlotCosMuMu.SetLineStyle(kDashed);

  signalTotalPlotmassPsiPi.SetLineColor(kRed); signalTotalPlotmassPsiPi.SetLineWidth(2);
  // signalTotalSigPlotmassPsiPi.SetLineColor(kRed); signalTotalSigPlotmassPsiPi.SetLineWidth(2); signalTotalSigPlotmassPsiPi.SetLineStyle(kDashDotted);
  // signalTotalBkgPlotmassPsiPi.SetLineColor(kRed); signalTotalBkgPlotmassPsiPi.SetLineWidth(2); signalTotalBkgPlotmassPsiPi.SetLineStyle(kDashed);

  signalTotalPlotPhi.SetLineColor(kRed); signalTotalPlotPhi.SetLineWidth(2);
  // signalTotalSigPlotPhi.SetLineColor(kRed); signalTotalSigPlotPhi.SetLineWidth(2); signalTotalSigPlotPhi.SetLineStyle(kDashDotted);
  // signalTotalBkgPlotPhi.SetLineColor(kRed); signalTotalBkgPlotPhi.SetLineWidth(2); signalTotalBkgPlotPhi.SetLineStyle(kDashed);

  //fptype totalIntegral = totalPdf->normalise();
  fptype totalIntegral = matrix->normalise();
  //fptype totalComponent = 0.;
  fptype compsIntegral = 0.0;
  std::cout <<"\nTotal Normalisation Factor = " <<totalIntegral <<std::endl;

  int kCounter = 0;
  Int_t nStatEntries = 0;
  Int_t amplitudeCounter = 0;
  for (size_t u=0; u<nKstars; ++u) {
    fitStat->AddText(TString::Format("\n------------------  %s  ------------------", kStarNames[kCounter].c_str()));
    ((TText*)fitStat->GetListOfLines()->Last())->SetTextColor(kCounter+3);
    addHelAmplStat(fitStat, "0", as[amplitudeCounter], bs[amplitudeCounter]); ++amplitudeCounter;
    nStatEntries +=2 ;

    if (Spins[u]->value > 0.) {
      addHelAmplStat(fitStat, "+1", as[amplitudeCounter], bs[amplitudeCounter]); ++amplitudeCounter;
      addHelAmplStat(fitStat, "-1", as[amplitudeCounter], bs[amplitudeCounter]); ++amplitudeCounter;
      nStatEntries +=2 ;
    }

    ++kCounter;
  }

  fitStat->SetTextAlign(12);
  fitStat->SetShadowColor(0);
  fitStat->SetFillColor(0);

  totalPdf->clearCurrentFit();

  legPlot->AddEntry(&massKPiHisto, "Generated data", "lpe");
  legPlot->AddEntry(&signalTotalPlotMKPi, "Total fit", "l");
  if(bkgPhaseSpace && !effPdfProd)
  {
    legPlot->AddEntry(&signalTotalBkgPlotMKPi, "Phase space only", "l");
    legPlot->AddEntry(&signalTotalSigPlotMKPi, "K* signal only", "l");
  }
  //multiGraphMKPi->Add(&signalTotalPlot,"L");
  ////////////////////////////////////////////////////////////////////////////////
  ///// COMPONENTS PDF PLOT
  ////////////////////////////////////////////////////////////////////////////////


  kCounter = 0;

  std::vector<Variable*> MassesPlot;
  std::vector<Variable*> GammasPlot;
  std::vector<Variable*> SpinsPlot;
  std::vector<Variable*> asPlot;
  std::vector<Variable*> bsPlot;

  std::vector<TH1F*> compHistosMKPi;
  std::vector<TH1F*> compHistosCosMuMu;
  std::vector<TH1F*> compHistosmassPsiPi;
  std::vector<TH1F*> compHistosPhi;

  Bool_t plotSingleKstars = kTRUE; //plotSingleKstars = kFALSE;

  int lastAmplitude = 0;

  for (int i = 0; i < nKstars; ++i) {
    MassesPlot.push_back(Masses[i]);
    GammasPlot.push_back(Gammas[i]);
    SpinsPlot.push_back(Spins[i]);
  }

  for (int k = 0; k < nKstars; ++k) {

    ////////////////////////////////////////////////////////////////////////////////
    //Initialising projection vectors

    std::vector<fptype> mkpCompProjection;
    std::vector<fptype> cosMuMuCompProjection;
    std::vector<fptype> massPsiPiCompProjection;
    std::vector<fptype> phiCompProjection;

    for (int i = 0; i < massKPi->numbins; ++i) {
      mkpCompProjection.push_back(0.0);
    }

    for (int i = 0; i < cosMuMu->numbins; ++i) {
      cosMuMuCompProjection.push_back(0.0);
    }

    for (int i = 0; i < massPsiPi->numbins; ++i) {
      massPsiPiCompProjection.push_back(0.0);
    }

    for (int i = 0; i < phi->numbins; ++i) {
      phiCompProjection.push_back(0.0);
    }

    ////////////////////////////////////////////////////////////////////////////////
    // Pushing histogram for each projection
    sprintf(bufferstring,"comp_%d_plotHisto_MKPi",kCounter);
    compHistosMKPi.push_back(new TH1F(bufferstring,bufferstring,massKPi->numbins, massKPi->lowerlimit, massKPi->upperlimit));
    sprintf(bufferstring,"comp_%d_plotHisto_CMM",kCounter);
    compHistosCosMuMu.push_back(new TH1F(bufferstring,bufferstring,cosMuMu->numbins, cosMuMu->lowerlimit, cosMuMu->upperlimit));
    sprintf(bufferstring,"comp_%d_plotHisto_CKS",kCounter);
    compHistosmassPsiPi.push_back(new TH1F(bufferstring,bufferstring,massPsiPi->numbins, massPsiPi->lowerlimit, massPsiPi->upperlimit));
    sprintf(bufferstring,"comp_%d_plotHisto_PHI",kCounter);
    compHistosPhi.push_back(new TH1F(bufferstring,bufferstring,phi->numbins, phi->lowerlimit, phi->upperlimit));


    cout <<"\n- Plotting " <<kStarNames[kCounter] <<" component by setting all other components to zero" <<endl;
    sum = 0.0;

    ////////////////////////////////////////////////////////////////////////////////
    // Setting other components to zero and fixing all useful parameters
    for (int l = 0; l < nHelAmps; ++l) {
      as[l]->fixed = true;
      bs[l]->fixed = true;
      //std::cout<<originalAs[l]<<" - "<<originalBs[l]<<std::endl;
    }

    for (int l = 0; l < nHelAmps; ++l) {
      asPlot.push_back(new Variable("zero_a",0.0));
      bsPlot.push_back(new Variable("zero_b",0.0));
    }

    // std::cout<<" Plotting KStars - Mass : "<<Masses[k]->value<<" Spin : "<<Spins[k]->value<<"Last Amplitude : "<<lastAmplitude<<std::endl;
    //For Spin = 0.0 only one component
    if (Spins[k]->value==0.0) {

      as[lastAmplitude]->fixed;
      bs[lastAmplitude]->fixed;
      // std::cout<<" - Amplitude vector pushing: "<<lastAmplitude<<" index ";
      // asPlot.push_back(as[lastAmplitude]);
      // bsPlot.push_back(bs[lastAmplitude]);
      asPlot[lastAmplitude]->value = originalAs[lastAmplitude];
      bsPlot[lastAmplitude]->value = originalBs[lastAmplitude];

      // for (int j = 0; j < nHelAmps; ++j) {
      //   if (j!=lastAmplitude) {
      //     // std::cout<<" putting zero: "<<j<<" index "<<std::endl;
      // 	  asPlot.push_back(new Variable("zero_a",0.0));
      // 	  bsPlot.push_back(new Variable("zero_b",0.0));
      //   }
      // }
      ++lastAmplitude;
    } else {
      // For Spin != 0 three components
      for (int i = lastAmplitude; i <= lastAmplitude+2; ++i) {
        // std::cout<<" - Amplitude vector pushing: "<<i<<" index ";
        as[i]->fixed;
        bs[i]->fixed;
        // asPlot.push_back(as[i]);
        // bsPlot.push_back(bs[i]);
        asPlot[i]->value = originalAs[i];
        bsPlot[i]->value = originalBs[i];
      }
  //     for (int d = 0; d < nHelAmps; ++d) {
	// if (d!=lastAmplitude && d!=lastAmplitude+1 && d!=lastAmplitude+2) {
	//   // std::cout<<" putting zero: "<<d<<" index "<<std::endl;
	//   asPlot.push_back(new Variable("zero_a",0.0));
	//   bsPlot.push_back(new Variable("zero_b",0.0));
	// }}
      lastAmplitude+=3;
    }

    // std::cout<<" --- "<<kCounter<<std::endl;
    // for (size_t i = 0; i < asPlot.size(); i++) {
    //   std::cout<<" - "<<i+1<<" A : "<<asPlot[i]->value<<" B : "<<bsPlot[i]->value<<std::endl;
    // }

    ////////////////////////////////////////////////////////////////////////////////
    // Normalising, integrating and evaluating the single component pdf

    sprintf(bufferstring,"Kstars_signal_plot_%d",kCounter);
    GooPdf* matrixPlot = new MatrixPdf(bufferstring, massKPi, cosMuMu, massPsiPi, phi,MassesPlot,GammasPlot,SpinsPlot,asPlot,bsPlot,psi_nS,dRadB0,dRadKs);
    matrixPlot->setData(&plottingGridData);

    matrixPlot->copyParams();
    compsIntegral = matrixPlot->normalise();

    fractions.push_back(compsIntegral/totalIntegral);
    //fractions.push_back(compsIntegral);

    cout <<"  Component " <<kStarNames[kCounter]<<" normalisation factor : " <<compsIntegral<<" (fraction: " <<compsIntegral/totalIntegral*100.0<<"%)" <<endl;

    matrixPlot->getCompProbsAtDataPoints(pdfCompValues);

    matrixPlot->clearCurrentFit();

    for (int k = 0; k<pdfCompValues[0].size();k++) {
      //std::cout <<" Bin : " << k << " pdf : " << pdfCompValues[0][k] <<std::endl;
      sum += pdfCompValues[0][k];
    }

    for (int k = 0; k<pdfCompValues[0].size();k++) {
      pdfCompValues[0][k] /=sum;
      pdfCompValues[0][k] *= events;
      pdfCompValues[0][k] *= (compsIntegral/totalIntegral);
      //compHistosMKPi[kCounter]->SetBinContent(k,pdfCompValues[0][k]);
    }

    ////////////////////////////////////////////////////////////////////////////////
    //Filling single components projections histos

    //MassKPi
    for (int j = 0; j < massKPi->numbins; ++j) {
      for (int i = 0; i < notMPKBins; ++i) {
	mkpCompProjection[j] += pdfCompValues[0][j+i*massKPi->numbins];
      }
      compHistosMKPi[kCounter]->SetBinContent(j+1,mkpCompProjection[j]);
    }

    compHistosMKPi[kCounter]->Scale(ratioMKPi);

    //Cos K Star
    for (int j = 0; j < cosMuMu->numbins; ++j) {
      for (int k = 0; k < phi->numbins; ++k) {
	for (int i = 0; i < massKPi->numbins*cosMuMu->numbins; ++i) {
	  cosMuMuCompProjection[j] += pdfCompValues[0][i+j*massKPi->numbins*massPsiPi->numbins+k*massKPi->numbins*cosMuMu->numbins*massPsiPi->numbins];
	}
      }
      compHistosCosMuMu[kCounter]->SetBinContent(j+1,cosMuMuCompProjection[j]);
    }

    compHistosCosMuMu[kCounter]->Scale(ratioCosMuMu);

    //Cos Mu Mu
    for (int j = 0; j < massPsiPi->numbins; ++j) {
      for (int k = 0; k < phi->numbins*cosMuMu->numbins; ++k) {
	for (int i = 0; i < massKPi->numbins; ++i) {
	  massPsiPiCompProjection[j] += pdfCompValues[0][i+k*massKPi->numbins*massPsiPi->numbins+j*massKPi->numbins];
	}
      }
      compHistosmassPsiPi[kCounter]->SetBinContent(j+1,massPsiPiCompProjection[j]);
    }

    compHistosmassPsiPi[kCounter]->Scale(ratioMassPsiPi);

    //Phi
    for (int j = 0; j < phi->numbins; ++j) {
      for (int k = 0; k < massPsiPi->numbins*massKPi->numbins*cosMuMu->numbins; ++k) {
	phiCompProjection[j] += pdfCompValues[0][k+j*massKPi->numbins*cosMuMu->numbins*massPsiPi->numbins];
      }
      compHistosPhi[kCounter]->SetBinContent(j+1,phiCompProjection[j]);
    }

    compHistosPhi[kCounter]->Scale(ratioPhi);

    if (nKstars > 1  &&  plotSingleKstars) {

      // Kstas components points array for each projection
      fptype pointsXCompMKPi[massKPi->numbins];
      fptype pointsYCompMKPi[massKPi->numbins];

      fptype pointsXCompCosMuMu[cosMuMu->numbins];
      fptype pointsYCompCosMuMu[cosMuMu->numbins];

      fptype pointsXCompmassPsiPi[massPsiPi->numbins];
      fptype pointsYCompmassPsiPi[massPsiPi->numbins];

      fptype pointsXCompPhi[phi->numbins];
      fptype pointsYCompPhi[phi->numbins];

      // Filling vectors for components projections graphs
      for (int k=0; k < massKPi->numbins; k++) {
        pointsXCompMKPi[k] = compHistosMKPi[kCounter]->GetBinCenter(k+1);
	pointsYCompMKPi[k] = compHistosMKPi[kCounter]->GetBinContent(k+1);
      }
      for (int k=0;k<cosMuMu->numbins;k++) {
      	pointsXCompCosMuMu[k] = compHistosCosMuMu[kCounter]->GetBinCenter(k+1);
      	pointsYCompCosMuMu[k] = compHistosCosMuMu[kCounter]->GetBinContent(k+1);
      }
      for (int k=0;k<massPsiPi->numbins;k++) {
      	pointsXCompmassPsiPi[k] = compHistosmassPsiPi[kCounter]->GetBinCenter(k+1);
      	pointsYCompmassPsiPi[k] = compHistosmassPsiPi[kCounter]->GetBinContent(k+1);
      }
      for (int k=0;k<phi->numbins;k++) {
      	pointsXCompPhi[k] = compHistosPhi[kCounter]->GetBinCenter(k+1);
      	pointsYCompPhi[k] = compHistosPhi[kCounter]->GetBinContent(k+1);
      }

      // Filling components projections graphs
      TGraph* signalCompPlotMKPi = new TGraph(massKPi->numbins, pointsXCompMKPi, pointsYCompMKPi);
      signalCompPlotMKPi->SetLineColor(kCounter+3);
      signalCompPlotMKPi->SetLineWidth(2);
      signalCompPlotMKPi->SetLineStyle(kDashed);
      signalCompPlotMKPi->GetXaxis()->SetTitle("m(K#Pi)");
      sprintf(bufferstring,"Events / (%.3f)",(massKPi->upperlimit - massKPi->lowerlimit)/massKPi->numbins);
      signalCompPlotMKPi->GetYaxis()->SetTitle(bufferstring);
      //signalCompPlotMKPi->Draw("");
      multiGraphMKPi->Add(signalCompPlotMKPi,"L");

      //
      TGraph* signalCompPlotCosMuMu = new TGraph(cosMuMu->numbins,pointsXCompCosMuMu,pointsYCompCosMuMu);
      signalCompPlotCosMuMu->SetLineColor(kCounter+3);
      signalCompPlotCosMuMu->SetLineWidth(2);
      signalCompPlotCosMuMu->SetLineStyle(kDashed);
      signalCompPlotCosMuMu->GetXaxis()->SetTitle("m(K#Pi)");
      sprintf(bufferstring,"Events / (%.3f)",(cosMuMu->upperlimit- cosMuMu->lowerlimit)/cosMuMu->numbins);
      signalCompPlotCosMuMu->GetYaxis()->SetTitle(bufferstring);
      //signalCompPlot->Draw("");
      multiGraphCosMuMu->Add(signalCompPlotCosMuMu,"L");

      TGraph* signalCompPlotmassPsiPi = new TGraph(massPsiPi->numbins,pointsXCompmassPsiPi,pointsYCompmassPsiPi);
      signalCompPlotmassPsiPi->SetLineColor(kCounter+3);
      signalCompPlotmassPsiPi->SetLineWidth(2);
      signalCompPlotmassPsiPi->SetLineStyle(kDashed);
      signalCompPlotmassPsiPi->GetXaxis()->SetTitle("m(K#Pi)");
      sprintf(bufferstring,"Events / (%.3f)",(massPsiPi->upperlimit- massPsiPi->lowerlimit)/massPsiPi->numbins);
      signalCompPlotmassPsiPi->GetYaxis()->SetTitle(bufferstring);
      //signalCompPlot->Draw("");
      multiGraphmassPsiPi->Add(signalCompPlotmassPsiPi,"L");

      TGraph* signalCompPlotPhi = new TGraph(phi->numbins,pointsXCompPhi,pointsYCompPhi);
      signalCompPlotPhi->SetLineColor(kCounter+3);
      signalCompPlotPhi->SetLineWidth(2);
      signalCompPlotPhi->SetLineStyle(kDashed);
      signalCompPlotPhi->GetXaxis()->SetTitle("m(K#Pi)");
      sprintf(bufferstring,"Events / (%.3f)",(phi->upperlimit- phi->lowerlimit)/phi->numbins);
      signalCompPlotPhi->GetYaxis()->SetTitle(bufferstring);
      //signalCompPlot->Draw("");
      multiGraphPhi->Add(signalCompPlotPhi,"L");

      sprintf(bufferstring,"%s (%.2f %)",kStarNames[kCounter].c_str(),compsIntegral/totalIntegral*100.);
      legPlot->AddEntry(signalCompPlotMKPi,bufferstring,"l");

    } // if (plotSingleKstars)

    /*
      massKPiHisto.Draw("");
      signalCompPlot->Draw("sameL");

      sprintf(bufferstring,"plots/plot%d.eps",kCounter);
      canvas->SetLogy(1);
      canvas->SaveAs(bufferstring);
      canvas->Clear();
    */
    ++kCounter;

    asPlot.clear();
    bsPlot.clear();
    pdfCompValues.clear();

  } // for (int k = 0; k < nHelAmps; ++k) {

  //Adding single points to plot better and total plots

  fptype pointsX[2], pointsY[2];
  pointsX[0] = massKPi->lowerlimit; pointsX[1] = massKPi->upperlimit;
  pointsY[0] = 0.01; pointsY[1] = massKPiHisto.GetMaximum();
  TGraph* pointsMKP = new TGraph(2,pointsX,pointsY);
  if(bkgPhaseSpace && !effPdfProd)
  {
    multiGraphMKPi->Add(&signalTotalBkgPlotMKPi,"L");
    //multiGraphMKPi->Add(&signalTotalSigPlotMKPi,"L");
  }
  multiGraphMKPi->Add(&signalTotalPlotMKPi,"L");
  //multiGraphMKPi->Add(pointsMKP,"P");

  pointsX[0] = massPsiPi->lowerlimit; pointsX[1] = massPsiPi->upperlimit;
  pointsY[0] = 0.01; pointsY[1] = massPsiPiHisto.GetMaximum();
  TGraph* pointsCKS = new TGraph(2,pointsX,pointsY);
  // if(bkgPhaseSpace)
  // {
  //   multiGraphmassPsiPi->Add(&signalTotalBkgPlotmassPsiPi,"L");
  //   multiGraphmassPsiPi->Add(&signalTotalSigPlotmassPsiPi,"L");
  // }
  multiGraphmassPsiPi->Add(&signalTotalPlotmassPsiPi,"L");
  multiGraphmassPsiPi->Add(pointsCKS,"P");

  pointsX[0] = cosMuMu->lowerlimit; pointsX[1] = cosMuMu->upperlimit;
  pointsY[0] = 0.01; pointsY[1] = cosMuMuHisto.GetMaximum();
  TGraph* pointsCMM = new TGraph(2,pointsX,pointsY);
  // if(bkgPhaseSpace)
  // {
  //   multiGraphCosMuMu->Add(&signalTotalBkgPlotCosMuMu,"L");
  //   multiGraphCosMuMu->Add(&signalTotalSigPlotCosMuMu,"L");
  // }
  multiGraphCosMuMu->Add(&signalTotalPlotCosMuMu,"L");
  multiGraphCosMuMu->Add(pointsCMM,"P");

  pointsX[0] = phi->lowerlimit; pointsX[1] = phi->upperlimit;
  pointsY[0] = 0.01; pointsY[1] = phiHisto.GetMaximum();
  TGraph* pointsPHI = new TGraph(2,pointsX,pointsY);
  // multiGraphPhi->Add(&signalTotalBkgPlotPhi,"L");
  // multiGraphPhi->Add(&signalTotalSigPlotPhi,"L");
  multiGraphPhi->Add(&signalTotalPlotPhi,"L");
  multiGraphPhi->Add(pointsPHI,"P");

  ////////////////////////////////////////////////////////////////////////////////
  // PLOTTING

  ////////////////////////////////////////////////////////////////////////////////
  // PLOTTING

  if (bkgPhaseSpaceMap) {
    bkgMKPi->Scale(events*bkgFrac);
    bkgMPsiPi->Scale(events*bkgFrac);
    bkgCMuMu->Scale(events*bkgFrac);
    bkgPhi->Scale(events*bkgFrac);

    fptype ratioMKPiBkg = ((fptype)(bkgMKPi->GetNbinsX()))/((fptype)datapoints1);
    fptype ratioCosMuMuBkg = (((fptype)(bkgMPsiPi->GetNbinsX()))/((fptype)datapoints2));
    fptype ratioMassPsiPiBkg = ((fptype)(bkgCMuMu->GetNbinsX()))/((fptype)datapoints3);
    fptype ratioPhiBkg = ((fptype)(bkgPhi->GetNbinsX()))/((fptype)datapoints4);

    bkgMKPi->Scale(ratioMKPiBkg);
    bkgMPsiPi->Scale(ratioMassPsiPiBkg);
    bkgCMuMu->Scale(ratioCosMuMuBkg);
    bkgPhi->Scale(ratioPhiBkg);
  }

  legPlot->SetY1( yMax - 0.05*(legPlot->GetNRows()) ) ;
  fitStat->SetY1( yMax - 0.03*nStatEntries ) ;
  //Mass K Pi
  multiGraphMKPi->Draw("AL");
  massKPiHisto.Draw("Esame");
  bkgMKPi->Draw("same");
  legPlot->Draw(); fitStat->Draw();

  // first Logy(1) and after Logy(0), viceversa does not work
  canvas->SetLogy(1);
  canvas->SaveAs(TString::Format("%s/%s%s__logy.%s",plotsDir.Data(),massKPi_name.Data(),plotsName.Data(),extension.Data()));
  canvas->SetLogy(0);
  canvas->SaveAs(TString::Format("%s/%s%s.%s",plotsDir.Data(),massKPi_name.Data(),plotsName.Data(),extension.Data()));
  canvas->Clear();
  ////////////////////////////////////////////////////////////////////////////////

  //CosMuMu
  multiGraphCosMuMu->Draw("AL");
  cosMuMuHisto.Draw("Esame");
  bkgCMuMu->Draw("same");
  // it's enough on the m(KPi) plot
  //legPlot->Draw(); //fitStat->Draw();

  canvas->SetLogy(1);
  canvas->SaveAs(TString::Format("%s/%s%s__logy.%s",plotsDir.Data(),cosMuMu_name.Data(),plotsName.Data(),extension.Data()));
  canvas->SetLogy(0);
  canvas->SaveAs(TString::Format("%s/%s%s.%s",plotsDir.Data(),cosMuMu_name.Data(),plotsName.Data(),extension.Data()));
  canvas->Clear();
  ////////////////////////////////////////////////////////////////////////////////

  //massPsiPi
  multiGraphmassPsiPi->Draw("AL");
  massPsiPiHisto.Draw("Esame");
  bkgMPsiPi->Draw("same");
  // it's enough on the m(KPi) plot
  //legPlot->Draw(); //fitStat->Draw();

  canvas->SetLogy(1);
  canvas->SaveAs(TString::Format("%s/%s%s__logy.%s",plotsDir.Data(),massPsiPi_name.Data(),plotsName.Data(),extension.Data()));
  canvas->SetLogy(0);
  canvas->SaveAs(TString::Format("%s/%s%s.%s",plotsDir.Data(),massPsiPi_name.Data(),plotsName.Data(),extension.Data()));
  canvas->Clear();
  ////////////////////////////////////////////////////////////////////////////////

  //Phi
  multiGraphPhi->Draw("AL");
  phiHisto.Draw("Esame");
  bkgPhi->Draw("same");
  // it's enough on the m(KPi) plot
  //legPlot->Draw(); //fitStat->Draw();

  canvas->SetLogy(1);
  canvas->SaveAs(TString::Format("%s/%s%s__logy.%s",plotsDir.Data(),phi_name.Data(),plotsName.Data(),extension.Data()));
  canvas->SetLogy(0);
  canvas->SaveAs(TString::Format("%s/%s%s.%s",plotsDir.Data(),phi_name.Data(),plotsName.Data(),extension.Data()));
  canvas->Clear();
  ////////////////////////////////////////////////////////////////////////////////

  cout <<endl;
  cout <<"~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~" <<endl;
  cout << "PDF fitting time:       " << (fitClocks / CLOCKS_PER_SEC) << " s" << endl ;
  cout << "Data plotting time:     " << (dataSetClocks / CLOCKS_PER_SEC) << " s" << endl ;
  cout << "PDF sum time:           " << (sumClocks / CLOCKS_PER_SEC) << " s" << endl ;
  cout << "PDF normalisation time: " << (normClocks / CLOCKS_PER_SEC) << " s" << endl ;
  cout << "PDF projection time:    " << (projClocks / CLOCKS_PER_SEC) << " s" << endl ;
  cout <<"~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~.~" <<endl;


  /*
    matrix->getCompProbsAtDataPoints(pdfTotalValuesNorm,events);
    for (int j = 0; j < massKPi->numbins; ++j) {
    projMKPiHisto.SetBinContent(j+1,mkpTotalProjection[j]);
    std::cout <<" Bin " <<j<<" center = " <<projMKPiHisto.GetBinCenter(j+1)<<" : " <<mkpTotalProjection[j]<<std::endl;
    }
  */

  /*
    UnbinnedDataSet tempData(obserVariables);

    std::vector<std::vector<fptype> > tempValues;

    massKPi->value = 1.0;
    massPsiPi->value = 0.5;
    cosMuMu->value = 0.5;
    phi->value = 0.25;

    tempData.addEvent();
    matrix->getCompProbsAtDataPoints(tempValues);

    std::cout << "Pdf value : " <<tempValues[0][0]<<std::endl;
  */

  return 0;

}
